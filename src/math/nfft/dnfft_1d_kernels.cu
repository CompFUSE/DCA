#include "hip/hip_runtime.h"
// Copyright (C) 2018 ETH Zurich
// Copyright (C) 2018 UT-Battelle, LLC
// All rights reserved.
//
// See LICENSE.txt for terms of usage.
// See CITATION.txt for citation guidelines if you use this code for scientific publications.
//
// Author: Giovanni Balduzzi (gbalduzz@itp.phys.ethz.ch)
//
// This file implements the GPU kernels used by the Dnfft1D class.

#include "dca/math/nfft/kernels_interface.hpp"

#include <array>
#include <hip/hip_runtime.h>

#include "dca/math/nfft/nfft_helper.cuh"
#include "dca/linalg/util/atomic_add_cuda.cu.hpp"
#include "dca/util/integer_division.hpp"

namespace dca {
namespace math {
namespace nfft {
namespace details {
// dca::math::nfft::details::

std::array<int, 2> getBlockSize(const int ni, const int block_size) {
  const int n_threads = std::min(block_size, ni);
  const int n_blocks = util::ceilDiv(ni, n_threads);
  return std::array<int, 2>{n_blocks, n_threads};
}

std::array<dim3, 2> getBlockSize(const uint i, const uint j, const uint block_size = 32) {
  const uint n_threads_i = std::min(block_size, i);
  const uint n_threads_j = std::min(block_size, j);
  if (n_threads_i * n_threads_j > 32 * 32)
    throw(std::logic_error("Block size is too big"));

  const uint n_blocks_i = dca::util::ceilDiv(i, n_threads_i);
  const uint n_blocks_j = dca::util::ceilDiv(j, n_threads_j);

  return std::array<dim3, 2>{dim3(n_blocks_i, n_blocks_j), dim3(n_threads_i, n_threads_j)};
}

// TODO: consider constant or texture memory for the coefficients.
template <int oversampling, int window_sampling, typename ScalarIn, typename ScalarOut,
          bool accumulate_m_sqr = false>
__global__ void accumulateOnDeviceKernel(
    const ScalarIn* __restrict__ M, const int ldm, const ScalarIn sign, ScalarOut* __restrict__ out,
    ScalarOut* __restrict__ out_sqr, int ldo, const ConfigElem* __restrict__ config_left,
    const ConfigElem* __restrict__ config_right, const ScalarIn* __restrict__ times,
    const ScalarOut* __restrict__ cubic_coeff, const int m_size) {
  constexpr int conv_size = 2 * oversampling;
  int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (thread_idx >= m_size * m_size * conv_size)
    return;

  // Unroll thread index.
  const int m_j = thread_idx / (m_size * conv_size);
  thread_idx -= m_j * (m_size * conv_size);
  const int m_i = thread_idx / conv_size;
  const int conv_idx = thread_idx - m_i * conv_size + 1;

  const ScalarOut tau = nfft_helper.computeTau(times[m_i], times[m_j]);

  int t_idx, conv_coeff_idx;
  ScalarOut delta_t;
  nfft_helper.computeInterpolationIndices<CUBIC, oversampling, window_sampling>(
      tau, t_idx, conv_coeff_idx, delta_t);

  const int linindex = nfft_helper.computeLinearIndex(
      config_left[m_i].band, config_right[m_j].band, config_left[m_i].site, config_right[m_j].site);

  const auto f_val = M[m_i + ldm * m_j];
  const auto* conv_coeff = cubic_coeff + conv_coeff_idx + 4 * conv_idx;
  ScalarOut* const out_ptr = out + t_idx + ldo * linindex + conv_idx;

  const auto conv_function_value =
      ((conv_coeff[3] * delta_t + conv_coeff[2]) * delta_t + conv_coeff[1]) * delta_t + conv_coeff[0];
  const auto contribution = f_val * sign * conv_function_value;
  linalg::atomicAdd(out_ptr, contribution);
  if (accumulate_m_sqr) {
    linalg::atomicAdd(out_sqr, f_val * f_val * conv_function_value);
  }
}

template <int oversampling, int window_sampling, typename ScalarIn, typename ScalarOut>
void accumulateOnDevice(const ScalarIn* M, const int ldm, const ScalarIn sign, ScalarOut* out,
                        ScalarOut* out_sqr, const int ldo, const ConfigElem* config_left,
                        const ConfigElem* config_right, const ScalarIn* tau,
                        const ScalarOut* cubic_coeff, const int size, hipStream_t stream_) {
  const auto blocks = getBlockSize(size * size * (2 * oversampling), 128);

  if (out_sqr) {
    accumulateOnDeviceKernel<oversampling, window_sampling, ScalarIn, ScalarOut, true>
        <<<blocks[0], blocks[1], 0, stream_>>>(M, ldm, sign, out, out_sqr, ldo, config_left,
                                               config_right, tau, cubic_coeff, size);
  }
  else {
    accumulateOnDeviceKernel<oversampling, window_sampling, ScalarIn, ScalarOut, false>
        <<<blocks[0], blocks[1], 0, stream_>>>(M, ldm, sign, out, out_sqr, ldo, config_left,
                                               config_right, tau, cubic_coeff, size);
  }
}

template <typename ScalarType>
__global__ void sumKernel(const ScalarType* in, const int ldi, ScalarType* out, const int ldo,
                          const int n, const int m) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  const int j = blockIdx.y * blockDim.y + threadIdx.y;

  if (i < n && j < m) {
    out[i + ldo * j] += in[i + ldi * j];
  }
}

template <typename ScalarType>
void sum(const ScalarType* in, const int ldi, ScalarType* out, const int ldo, const int n,
         const int m, hipStream_t stream) {
  auto blocks = getBlockSize(n, m, 16);
  sumKernel<<<blocks[0], blocks[1], 0, stream>>>(in, ldi, out, ldo, n, m);
}

void initializeNfftHelper(int nb, int nc, const int* add_r, int lda, const int* sub_r, int lds,
                          double t0, double delta_t, double t0_window, double delta_t_window,
                          double beta) {
  NfftHelper::set(nb, nc, add_r, lda, sub_r, lds, t0, delta_t, t0_window, delta_t_window, beta);
}

// Explicit instantiation.
constexpr int oversampling = 8;
constexpr int window_sampling = 32;
template void accumulateOnDevice<oversampling, window_sampling, double, double>(
    const double* M, const int ldm, const double sign, double* out, double* out_sqr, const int ldo,
    const ConfigElem* config_left, const ConfigElem* config_right, const double* tau,
    const double* cubic_coeff, const int size, hipStream_t stream_);
template void accumulateOnDevice<oversampling, window_sampling, float, float>(
    const float* M, const int ldm, const float sign, float* out, float* out_sqr, const int ldo,
    const ConfigElem* config_left, const ConfigElem* config_right, const float* tau,
    const float* cubic_coeff, const int size, hipStream_t stream_);

template void sum<double>(const double* in, const int ldi, double* out, const int ldo, const int n,
                          const int m, hipStream_t stream);
template void sum<float>(const float* in, const int ldi, float* out, const int ldo, const int n,
                         const int m, hipStream_t stream);

}  // namespace details
}  // namespace nfft
}  // namespace math
}  // namespace dca
