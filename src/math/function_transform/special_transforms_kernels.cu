#include "hip/hip_runtime.h"
// Copyright (C) 2018 ETH Zurich
// Copyright (C) 2018 UT-Battelle, LLC
// All rights reserved.
//
// See LICENSE.txt for terms of usage.
// See CITATION.txt for citation guidelines if you use this code for scientific publications.
//
// Author: Giovanni Balduzzi (gbalduzz@itp.phys.ethz.ch)
//
// Implements the GPU kernels used by the SpaceTransform2DGpu class.

#include "dca/math/function_transform/special_transforms/kernels_interface.hpp"

#include <array>

#include "dca/util/integer_division.hpp"
#include "dca/linalg/util/cast_cuda.hpp"
#include "dca/linalg/util/complex_operators_cuda.cu.hpp"

namespace dca {
namespace math {
namespace transform {
namespace details {
// dca::math::transform::details::

using linalg::util::CudaComplex;
using linalg::util::castCudaComplex;

std::array<dim3, 2> getBlockSize(const int i, const int j) {
  const int n_threads_i = std::min(32, i);
  const int n_threads_j = std::min(32, j);
  const int n_blocks_i = util::ceilDiv(i, n_threads_i);
  const int n_blocks_j = util::ceilDiv(j, n_threads_j);

  return std::array<dim3, 2>{dim3(n_blocks_i, n_blocks_j), dim3(n_threads_i, n_threads_j)};
}

template <typename Real>
__global__ void phaseFactorsAndRearrangeKernel(const CudaComplex<Real>* in, const int ldi,
                                               CudaComplex<Real>* out, const int ldo, const int nb,
                                               const int nk, const int nw,
                                               const CudaComplex<Real>* phase_factors) {
  const int id_i = blockIdx.x * blockDim.x + threadIdx.x;
  const int id_j = blockIdx.y * blockDim.y + threadIdx.y;

  const int cols = nb * nk * nw;
  const int rows = cols / 2;
  if (id_i >= rows || id_j >= cols)
    return;

  const int no = nb * nk;
  auto get_indices = [nk, no](int id, int& b, int& k, int& w) {
    w = id / no;
    id -= w * no;
    b = id / nk;
    k = id - b * nk;
  };
  int w1, w2, b1, b2, k1, k2;

  get_indices(id_i, b1, k1, w1);
  get_indices(id_j, b2, k2, w2);

  const int out_i = b1 + nb * k1 + no * w1;
  const int out_j = b2 + nb * k2 + no * w2;

  CudaComplex<Real> value = in[id_i + ldi * id_j];

  using namespace dca::linalg;
  if (phase_factors)
    value *= phase_factors[b1 + nb * k1] * linalg::conj(phase_factors[b2 + nb * k2]);

  out[out_i + ldo * out_j] = value;
}

template <typename Real>
void phaseFactorsAndRearrange(const std::complex<Real>* in, const int ldi, std::complex<Real>* out,
                              const int ldo, const int nb, const int nk, const int nw,
                              const std::complex<Real>* phase_factors, const hipStream_t stream) {
  const int size = nk * nb * nw;
  auto const blocks = getBlockSize(size / 2, size);

  phaseFactorsAndRearrangeKernel<Real>
      <<<blocks[0], blocks[1], 0, stream>>>(castCudaComplex(in), ldi, castCudaComplex(out), ldo, nb,
                                            nk, nw, castCudaComplex(phase_factors));
}

// Explicit instantiation.
template void phaseFactorsAndRearrange<double>(const std::complex<double>* in, const int ldi,
                                               std::complex<double>* out, const int ldo,
                                               const int nb, const int nk, const int nw,
                                               const std::complex<double>* phase_factors,
                                               const hipStream_t stream);
template void phaseFactorsAndRearrange<float>(const std::complex<float>* in, const int ldi,
                                              std::complex<float>* out, const int ldo, const int nb,
                                              const int nk, const int nw,
                                              const std::complex<float>* phase_factors,
                                              hipStream_t const stream);

}  // details
}  // transform
}  // math
}  // dca
