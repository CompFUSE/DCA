#include "hip/hip_runtime.h"
// Copyright (C) 2018 ETH Zurich
// Copyright (C) 2018 UT-Battelle, LLC
// All rights reserved.
//
// See LICENSE for terms of usage.
// See CITATION.md for citation guidelines, if DCA++ is used for scientific publications.
//
// Author: Giovanni Balduzzi (gbalduzz@itp.phys.ethz.ch)
//         Raffaele Solca' (rasolca@itp.phys.ethz.ch)
//
// This file implements laset_gpu.hpp.

#include "dca/linalg/lapack/multiply_diagonal_gpu.hpp"
#include <cassert>
#include "dca/platform/dca_gpu_complex.h"
#include "dca/platform/dca_gpu.h"
#include "dca/linalg/util/stream_functions.hpp"
#include "dca/util/integer_division.hpp"

namespace dca {
namespace linalg {
namespace lapack {
namespace kernels {
// dca::linalg::lapack::kernels::

constexpr int multiply_diag_block_size_x = 32;
constexpr int multiply_diag_block_size_y = 32;

template <typename ScalarIn, typename ScalarOut>
__global__ void multiplyDiagonalLeft(int m, int n, const ScalarIn* d, int inc_d, const ScalarIn* a,
                                     int lda, ScalarOut* b, int ldb) {
  const int i = threadIdx.x + blockIdx.x * blockDim.x;
  const int j = threadIdx.y + blockIdx.y * blockDim.y;

  if (i < m && j < n) {
    b[i + j * ldb] = d[i * inc_d] * a[i + j * lda];
  }
}

template <typename Type>
__global__ void multiplyDiagonalRight(int m, int n, const Type* a, int lda, const Type* d,
                                      int inc_d, Type* b, int ldb) {
  const int i = threadIdx.x + blockIdx.x * blockDim.x;
  const int j = threadIdx.y + blockIdx.y * blockDim.y;

  if (i < m && j < n) {
    b[i + j * ldb] = d[j * inc_d] * a[i + j * lda];
  }
}

}  // namespace kernels
// dca::linalg::lapack::

template <typename ScalarIn, typename ScalarOut>
void multiplyDiagonalLeft_gpu(int m, int n, const ScalarIn* d, int inc_d, const ScalarIn* a,
                              int lda, ScalarOut* b, int ldb, int thread_id, int stream_id) {
  assert(lda >= m);
  assert(ldb >= m);

  if (m > 0 && n > 0) {
    checkErrorsCudaDebug();
    const int bl_x = dca::util::ceilDiv(m, kernels::multiply_diag_block_size_x);
    const int bl_y = dca::util::ceilDiv(n, kernels::multiply_diag_block_size_y);

    const dim3 threads(kernels::multiply_diag_block_size_x, kernels::multiply_diag_block_size_y);
    const dim3 blocks(bl_x, bl_y);

    hipStream_t stream = dca::linalg::util::getStream(thread_id, stream_id);

    kernels::multiplyDiagonalLeft<ScalarIn, ScalarOut>
        <<<blocks, threads, 0, stream>>>(m, n, d, inc_d, a, lda, b, ldb);
    checkErrorsCudaDebug();
  }
}

template void multiplyDiagonalLeft_gpu<float, float>(int m, int n, const float* d, int inc_d,
                                                     const float* a, int lda, float* b, int ldb,
                                                     int thread_id, int stream_id);
template void multiplyDiagonalLeft_gpu<double, double>(int m, int n, const double* d, int inc_d,
                                                       const double* a, int lda, double* b, int ldb,
                                                       int thread_id, int stream_id);
template void multiplyDiagonalLeft_gpu<double, float>(int m, int n, const double* d, int inc_d,
                                                      const double* a, int lda, float* b, int ldb,
                                                      int thread_id, int stream_id);
template void multiplyDiagonalLeft_gpu<hipComplex, hipComplex>(int m, int n, const hipComplex* d,
                                                             int inc_d, const hipComplex* a, int lda,
                                                             hipComplex* b, int ldb, int thread_id,
                                                             int stream_id);
template void multiplyDiagonalLeft_gpu<hipDoubleComplex, hipDoubleComplex>(
    int m, int n, const hipDoubleComplex* d, int inc_d, const hipDoubleComplex* a, int lda,
    hipDoubleComplex* b, int ldb, int thread_id, int stream_id);

template <typename Type>
void multiplyDiagonalRight_gpu(int m, int n, const Type* a, int lda, const Type* d, int inc_d,
                               Type* b, int ldb, int thread_id, int stream_id) {
  assert(lda >= m);
  assert(ldb >= m);

  if (m > 0 && n > 0) {
    checkErrorsCudaDebug();
    const int bl_x = dca::util::ceilDiv(m, kernels::multiply_diag_block_size_x);
    const int bl_y = dca::util::ceilDiv(n, kernels::multiply_diag_block_size_y);

    const dim3 threads(kernels::multiply_diag_block_size_x, kernels::multiply_diag_block_size_y);
    const dim3 blocks(bl_x, bl_y);

    hipStream_t stream = dca::linalg::util::getStream(thread_id, stream_id);

    kernels::multiplyDiagonalRight<<<blocks, threads, 0, stream>>>(m, n, a, lda, d, inc_d, b, ldb);
    checkErrorsCudaDebug();
  }
}
template void multiplyDiagonalRight_gpu(int m, int n, const float* a, int lda, const float* d,
                                        int inc_d, float* b, int ldb, int thread_id, int stream_id);
template void multiplyDiagonalRight_gpu(int m, int n, const double* a, int lda, const double* d,
                                        int inc_d, double* b, int ldb, int thread_id, int stream_id);
template void multiplyDiagonalRight_gpu(int m, int n, const hipComplex* a, int lda,
                                        const hipComplex* d, int inc_d, hipComplex* b, int ldb,
                                        int thread_id, int stream_id);
template void multiplyDiagonalRight_gpu(int m, int n, const hipDoubleComplex* a, int lda,
                                        const hipDoubleComplex* d, int inc_d, hipDoubleComplex* b,
                                        int ldb, int thread_id, int stream_id);

}  // namespace lapack
}  // namespace linalg
}  // namespace dca
