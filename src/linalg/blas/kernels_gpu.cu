#include "hip/hip_runtime.h"
// Copyright (C) 2018 ETH Zurich
// Copyright (C) 2018 UT-Battelle, LLC
// All rights reserved.
//
// See LICENSE for terms of usage.
// See CITATION.md for citation guidelines, if DCA++ is used for scientific publications.
//
// Author: Giovanni Balduzzi (gbalduzz@itp.phys.ethz.ch)
//         Peter Staar (taa@zurich.ibm.com)
//         Raffaele Solca' (rasolca@itp.phys.ethz.ch)
//
// This file implements the kernels used by matrixop.hpp. See the latter for documentation.

#include "dca/linalg/blas/kernels_gpu.hpp"
#include <cassert>
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include "dca/linalg/util/complex_operators_cuda.cu.hpp"
#include "dca/linalg/util/error_cuda.hpp"
#include "dca/linalg/util/stream_functions.hpp"
#include "dca/util/integer_division.hpp"

namespace dca {
namespace linalg {
namespace blas {
namespace kernels {
// dca::linalg::blas::kernels::

constexpr int copy_col_block_size = 128;
constexpr int move_block_size_x = 32;
constexpr int move_block_size_y = 8;
constexpr int scale_block_size_x = 32;
constexpr int swap_block_size_x = 32;
constexpr int swap_block_size_y = 32;

template <typename Type>
__global__ void copyRows(int row_size, int n_rows, const int* i_x, const Type* x, int ldx,
                         const int* i_y, Type* y, int ldy) {
  // Work on BlockDim.x rows and copyrows_block_size_y cols.
  int ind_i = threadIdx.x + blockIdx.x * blockDim.x;
  int ind_j = threadIdx.y + blockIdx.y * blockDim.y;

  if (ind_i < n_rows && ind_j < row_size) {
    y[i_y[ind_i] + ind_j * ldy] = x[i_x[ind_i] + ind_j * ldx];
  }
}

template <typename Type>
__global__ void copyRows(int row_size, int n_rows, const int* i_x, const Type* x, int ldx, Type* y,
                         int ldy) {
  // Work on BlockDim.x rows and copyrows_block_size_y cols.
  int ind_i = threadIdx.x + blockIdx.x * blockDim.x;
  int ind_j = threadIdx.y + blockIdx.y * blockDim.y;

  if (ind_i < n_rows && ind_j < row_size) {
    y[ind_i + ind_j * ldy] = x[i_x[ind_i] + ind_j * ldx];
  }
}

template <typename Type>
__global__ void copyCols(int col_size, const int* j_x, const Type* x, int ldx, const int* j_y,
                         Type* y, int ldy) {
  // Each block copies a column.
  const int col_in_start = j_x[blockIdx.x] * ldx;
  const int col_out_start = j_y[blockIdx.x] * ldy;
  constexpr int stride = copy_col_block_size;

  // Coalesced memory access:
  for (int i = threadIdx.x; i < col_size; i += stride)
    y[i + col_out_start] = x[i + col_in_start];
}

template <typename Type>
__global__ void copyCols(int col_size, const int* j_x, const Type* x, int ldx, Type* y, int ldy) {
  // Each block copies a column.
  const int col_in_start = j_x[blockIdx.x] * ldx;
  const int col_out_start = blockIdx.x * ldy;
  constexpr int stride = copy_col_block_size;

  // Coalesced memory access:
  for (int i = threadIdx.x; i < col_size; i += stride)
    y[i + col_out_start] = x[i + col_in_start];
}

template <typename Type>
__global__ void moveLeft(int m, int n, Type* a, int lda) {
  assert(blockDim.y == 1);
  assert(blockDim.z == 1);
  assert(blockIdx.y == 0);
  assert(blockIdx.z == 0);

  int i = threadIdx.x + blockIdx.x * blockDim.x;

  if (i < m) {
    for (int j = 0; j < n - 1; ++j)
      a[i + j * lda] = a[i + (j + 1) * lda];
  }
}

template <typename Type>
__global__ void moveUp(int m, int n, Type* a, int lda) {
  assert(blockDim.x == move_block_size_x);
  assert(blockDim.y == 1);
  assert(blockDim.z == 1);
  assert(blockIdx.x == 0);
  assert(blockIdx.z == 0);

  __shared__ Type work[move_block_size_x * move_block_size_y];
  int ldw = move_block_size_x;

  int idx = threadIdx.x;
  int js = blockIdx.y * move_block_size_y;
  int je = min(n, (blockIdx.y + 1) * move_block_size_y);

  int jd = je - js;
  a += lda * js;

  int m_div = (m - 1) / blockDim.x * blockDim.x;

  for (int i = 0; i < m_div; i += blockDim.x) {
    for (int j = 0; j < jd; ++j)
      work[idx + ldw * j] = a[i + 1 + idx + lda * j];
    __syncthreads();
    for (int j = 0; j < jd; ++j)
      a[i + idx + lda * j] = work[idx + ldw * j];
    __syncthreads();
  }
  int i = m_div;
  if (i + idx < m - 1) {
    for (int j = 0; j < jd; ++j)
      work[idx + ldw * j] = a[i + 1 + idx + lda * j];
    __syncthreads();
    for (int j = 0; j < jd; ++j)
      a[i + idx + lda * j] = work[idx + ldw * j];
  }
}

template <typename Type>
__global__ void scaleRows(int row_size, int n_rows, const int* i, const Type* alpha, Type* a,
                          int lda) {
  const int ind_i = threadIdx.x + blockIdx.x * blockDim.x;
  const int j = threadIdx.y + blockIdx.y * blockDim.y;

  if (ind_i < n_rows && j < row_size) {
    a[i[ind_i] + j * lda] *= alpha[ind_i];
  }
}

template <typename Type>
__global__ void swapRows(int row_size, int n_rows, const int* i1, const int* i2, Type* a, int lda) {
  const int ind_i = threadIdx.x + blockIdx.x * blockDim.x;
  const int ind_j = threadIdx.y + blockIdx.y * blockDim.y;

  if (ind_i < n_rows && ind_j < row_size) {
    const Type tmp = a[i1[ind_i] + ind_j * lda];
    a[i1[ind_i] + ind_j * lda] = a[i2[ind_i] + ind_j * lda];
    a[i2[ind_i] + ind_j * lda] = tmp;
  }
}

template <typename Type>
__global__ void swapCols(int col_size, int n_cols, const int* j1, const int* j2, Type* a, int lda) {
  const int ind_i = threadIdx.x + blockIdx.x * blockDim.x;
  const int ind_j = threadIdx.y + blockIdx.y * blockDim.y;

  if (ind_i < col_size && ind_j < n_cols) {
    const Type tmp = a[ind_i + j1[ind_j] * lda];
    a[ind_i + j1[ind_j] * lda] = a[ind_i + j2[ind_j] * lda];
    a[ind_i + j2[ind_j] * lda] = tmp;
  }
}

}  // namespace kernels
// dca::linalg::blas::

template <typename Type>
void copyRows(int row_size, int n_rows, const int* i_x, const Type* x, int ldx, const int* i_y,
              Type* y, int ldy, int thread_id, int stream_id) {
  if (row_size > 0 && n_rows > 0) {
    checkErrorsCudaDebug();
    constexpr int block_size = 32;
    const int threads_x = std::min(block_size, n_rows);
    const int bl_x = dca::util::ceilDiv(n_rows, threads_x);
    int bl_y = dca::util::ceilDiv(row_size, block_size);

    const dim3 threads(threads_x, block_size);
    const dim3 blocks(bl_x, bl_y);

    hipStream_t stream = dca::linalg::util::getStream(thread_id, stream_id);

    kernels::copyRows<<<blocks, threads, 0, stream>>>(row_size, n_rows, i_x, x, ldx, i_y, y, ldy);
    checkErrorsCudaDebug();
  }
}
template void copyRows(int row_size, int n_rows, const int* i_x, const float* x, int ldx,
                       const int* i_y, float* y, int ldy, int thread_id, int stream_id);
template void copyRows(int row_size, int n_rows, const int* i_x, const double* x, int ldx,
                       const int* i_y, double* y, int ldy, int thread_id, int stream_id);
template void copyRows(int row_size, int n_rows, const int* i_x, const hipComplex* x, int ldx,
                       const int* i_y, hipComplex* y, int ldy, int thread_id, int stream_id);
template void copyRows(int row_size, int n_rows, const int* i_x, const hipDoubleComplex* x, int ldx,
                       const int* i_y, hipDoubleComplex* y, int ldy, int thread_id, int stream_id);

template <typename Type>
void copyRows(int row_size, int n_rows, const int* i_x, const Type* x, int ldx, Type* y, int ldy,
              int thread_id, int stream_id) {
  if (row_size > 0 && n_rows > 0) {
    checkErrorsCudaDebug();
    constexpr int block_size = 32;
    const int threads_x = std::min(block_size, n_rows);
    const int bl_x = dca::util::ceilDiv(n_rows, threads_x);
    int bl_y = dca::util::ceilDiv(row_size, block_size);

    const dim3 threads(threads_x, block_size);
    const dim3 blocks(bl_x, bl_y);

    hipStream_t stream = dca::linalg::util::getStream(thread_id, stream_id);

    kernels::copyRows<<<blocks, threads, 0, stream>>>(row_size, n_rows, i_x, x, ldx, y, ldy);
    checkErrorsCudaDebug();
  }
}

template void copyRows(int row_size, int n_rows, const int* i_x, const float* x, int ldx, float* y,
                       int ldy, int thread_id, int stream_id);
template void copyRows(int row_size, int n_rows, const int* i_x, const double* x, int ldx,
                       double* y, int ldy, int thread_id, int stream_id);

template <typename Type>
void copyCols(int col_size, int n_cols, const int* j_x, const Type* x, int ldx, const int* j_y,
              Type* y, int ldy, int thread_id, int stream_id) {
  if (col_size > 0 && n_cols > 0) {
    checkErrorsCudaDebug();
    hipStream_t stream = dca::linalg::util::getStream(thread_id, stream_id);

    kernels::copyCols<<<n_cols, kernels::copy_col_block_size, 0, stream>>>(col_size, j_x, x, ldx,
                                                                           j_y, y, ldy);
    checkErrorsCudaDebug();
  }
}
template void copyCols(int col_size, int n_cols, const int* j_x, const float* x, int ldx,
                       const int* j_y, float* y, int ldy, int thread_id, int stream_id);
template void copyCols(int col_size, int n_cols, const int* j_x, const double* x, int ldx,
                       const int* j_y, double* y, int ldy, int thread_id, int stream_id);
template void copyCols(int col_size, int n_cols, const int* j_x, const hipComplex* x, int ldx,
                       const int* j_y, hipComplex* y, int ldy, int thread_id, int stream_id);
template void copyCols(int col_size, int n_cols, const int* j_x, const hipDoubleComplex* x, int ldx,
                       const int* j_y, hipDoubleComplex* y, int ldy, int thread_id, int stream_id);

template <typename Type>
void copyCols(int col_size, int n_cols, const int* j_x, const Type* x, int ldx, Type* y, int ldy,
              int thread_id, int stream_id) {
  if (col_size > 0 && n_cols > 0) {
    checkErrorsCudaDebug();
    hipStream_t stream = dca::linalg::util::getStream(thread_id, stream_id);

    kernels::copyCols<<<n_cols, kernels::copy_col_block_size, 0, stream>>>(col_size, j_x, x, ldx, y,
                                                                           ldy);
    checkErrorsCudaDebug();
  }
}

template void copyCols(int, int, const int*, const float*, int, float*, int, int, int);
template void copyCols(int, int, const int*, const double*, int, double*, int, int, int);

template <typename Type>
void moveLeft(int m, int n, Type* a, int lda) {
  assert(lda >= m);

  if (m > 0 && n > 1) {
    checkErrorsCudaDebug();
    int bl_x = dca::util::ceilDiv(m, kernels::move_block_size_x);

    dim3 threads(kernels::move_block_size_x);
    dim3 blocks(bl_x);

    kernels::moveLeft<<<blocks, threads>>>(m, n, a, lda);
    checkErrorsCudaDebug();
  }
}
template void moveLeft(int m, int n, float* a, int lda);
template void moveLeft(int m, int n, double* a, int lda);
template void moveLeft(int m, int n, hipComplex* a, int lda);
template void moveLeft(int m, int n, hipDoubleComplex* a, int lda);

template <typename Type>
void moveUp(int m, int n, Type* a, int lda) {
  assert(lda >= m);

  if (m > 1 && n > 0) {
    checkErrorsCudaDebug();
    int bl_y = dca::util::ceilDiv(n, kernels::move_block_size_y);

    dim3 threads(kernels::move_block_size_x);
    dim3 blocks(1, bl_y);

    kernels::moveUp<<<blocks, threads>>>(m, n, a, lda);
    checkErrorsCudaDebug();
  }
}
template void moveUp(int m, int n, float* a, int lda);
template void moveUp(int m, int n, double* a, int lda);
template void moveUp(int m, int n, hipComplex* a, int lda);
template void moveUp(int m, int n, hipDoubleComplex* a, int lda);

template <typename Type>
void scaleRows(int row_size, int n_rows, const int* i, const Type* alpha, Type* a, int lda,
               int thread_id, int stream_id) {
  if (row_size > 0 && n_rows > 0) {
    checkErrorsCudaDebug();

    const int threads_x = std::min(kernels::scale_block_size_x, n_rows);
    const int threads_y = 1024 / threads_x;
    const dim3 threads(threads_x, threads_y);

    const int bl_x = dca::util::ceilDiv(n_rows, threads_x);
    const int bl_y = dca::util::ceilDiv(row_size, threads_y);

    const dim3 blocks(bl_x, bl_y);

    hipStream_t stream = dca::linalg::util::getStream(thread_id, stream_id);

    kernels::scaleRows<<<blocks, threads, 0, stream>>>(row_size, n_rows, i, alpha, a, lda);
    checkErrorsCudaDebug();
  }
}
template void scaleRows(int row_size, int n_rows, const int* i, const float* alpha, float* a,
                        int lda, int thread_id, int stream_id);
template void scaleRows(int row_size, int n_rows, const int* i, const double* alpha, double* a,
                        int lda, int thread_id, int stream_id);
template void scaleRows(int row_size, int n_rows, const int* i, const hipComplex* alpha,
                        hipComplex* a, int lda, int thread_id, int stream_id);
template void scaleRows(int row_size, int n_rows, const int* i, const hipDoubleComplex* alpha,
                        hipDoubleComplex* a, int lda, int thread_id, int stream_id);

template <typename Type>
void swapRows(int row_size, int n_rows, const int* i1, const int* i2, Type* a, int lda,
              int thread_id, int stream_id) {
  if (row_size > 0 && n_rows > 0) {
    const int threads_x = std::min(kernels::swap_block_size_x, n_rows);
    const int threads_y = 1024 / threads_x;
    const dim3 threads(threads_x, threads_y);

    const int bl_x = dca::util::ceilDiv(n_rows, threads_x);
    const int bl_y = dca::util::ceilDiv(row_size, threads_y);

    const dim3 blocks(bl_x, bl_y);

    hipStream_t stream = dca::linalg::util::getStream(thread_id, stream_id);

    kernels::swapRows<<<blocks, threads, 0, stream>>>(row_size, n_rows, i1, i2, a, lda);
    checkErrorsCudaDebug();
  }
}
template void swapRows(int row_size, int n_rows, const int* i1, const int* i2, float* a, int lda,
                       int thread_id, int stream_id);
template void swapRows(int row_size, int n_rows, const int* i1, const int* i2, double* a, int lda,
                       int thread_id, int stream_id);
template void swapRows(int row_size, int n_rows, const int* i1, const int* i2, hipComplex* a,
                       int lda, int thread_id, int stream_id);
template void swapRows(int row_size, int n_rows, const int* i1, const int* i2, hipDoubleComplex* a,
                       int lda, int thread_id, int stream_id);

template <typename Type>
void swapCols(int col_size, int n_cols, const int* j1, const int* j2, Type* a, int lda,
              int thread_id, int stream_id) {
  if (col_size > 0 && n_cols > 0) {
    checkErrorsCudaDebug();
    const int bl_x = dca::util::ceilDiv(col_size, kernels::swap_block_size_x);
    const int bl_y = dca::util::ceilDiv(n_cols, kernels::swap_block_size_y);

    dim3 threads(kernels::swap_block_size_x, kernels::swap_block_size_y);
    dim3 blocks(bl_x, bl_y);

    hipStream_t stream = dca::linalg::util::getStream(thread_id, stream_id);

    kernels::swapCols<<<blocks, threads, 0, stream>>>(col_size, n_cols, j1, j2, a, lda);
    checkErrorsCudaDebug();
  }
}
template void swapCols(int col_size, int n_cols, const int* j1, const int* j2, float* a, int lda,
                       int thread_id, int stream_id);
template void swapCols(int col_size, int n_cols, const int* j1, const int* j2, double* a, int lda,
                       int thread_id, int stream_id);
template void swapCols(int col_size, int n_cols, const int* j1, const int* j2, hipComplex* a,
                       int lda, int thread_id, int stream_id);
template void swapCols(int col_size, int n_cols, const int* j1, const int* j2, hipDoubleComplex* a,
                       int lda, int thread_id, int stream_id);
}  // namespace blas
}  // namespace linalg
}  // namespace dca
