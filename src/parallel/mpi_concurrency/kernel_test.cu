#include "hip/hip_runtime.h"
// Copyright (C) 2018 ETH Zurich
// Copyright (C) 2018 UT-Battelle, LLC
// All rights reserved.
//
// See LICENSE.txt for terms of usage.
// See CITATION.txt for citation guidelines if you use this code for scientific publications.
//
// Author: Giovanni Balduzzi (gbalduzz@itp.phys.ethz.ch)
//
// Test the execution of a GPU kernel.

#include "dca/parallel/mpi_concurrency/kernel_test.hpp"

#include <array>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "dca/linalg/util/error_cuda.hpp"

__global__ void kernel(int* out) {
  out[threadIdx.x] = threadIdx.x;
}

namespace dca {
namespace parallel {
// dca::parallel::

bool kernelTest() {
  int* dev;
  std::array<int, 32> host;

  hipMalloc(&dev, sizeof(int) * 32);
  kernel<<<1, 32>>>(dev);
  dca::linalg::util::checkErrorsCudaDebugInternal(__builtin_FUNCTION(), "kernel_test", 34);
  hipMemcpy(host.data(), dev, sizeof(int) * 32, hipMemcpyDeviceToHost);
  hipFree(dev);

  for (int i = 0; i < 32; ++i)
    if (host[i] != i)
      return false;

  return hipSuccess == hipPeekAtLastError();
}

}  // parallel
}  // dca
