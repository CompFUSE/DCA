#include "hip/hip_runtime.h"
// Copyright (C) 2018 ETH Zurich
// Copyright (C) 2018 UT-Battelle, LLC
// All rights reserved.
//
// See LICENSE.txt for terms of usage.
// See CITATION.txt for citation guidelines if you use this code for scientific publications.
//
// Author: Giovanni Balduzzi (gbalduzz@itp.phys.ethz.ch)
//
// Implements the GPU kernels used by 'cached_ndft_gpu.hpp'.

#include "dca/phys/dca_step/cluster_solver/shared_tools/accumulation/tp/ndft/kernels_interface.hpp"

#include <array>
#include <cassert>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "dca/util/integer_division.hpp"
#include "dca/linalg/util/cast_cuda.hpp"

namespace dca {
namespace phys {
namespace solver {
namespace accumulator {
namespace details {

using linalg::util::castCudaComplex;
using linalg::util::CudaComplex;

std::array<dim3, 2> getBlockSize(const int i, const int j) {
  assert(i > 0 && j > 0);
  const int n_threads_i = std::min(32, i);
  const int n_threads_j = std::min(32, j);
  const int n_blocks_i = util::ceilDiv(i, n_threads_i);
  const int n_blocks_j = util::ceilDiv(j, n_threads_j);

  return std::array<dim3, 2>{dim3(n_blocks_i, n_blocks_j), dim3(n_threads_i, n_threads_j)};
}

template <typename InpScalar, typename Real>
__global__ void sortMKernel(const int size, const InpScalar* M, const int ldm,
                            CudaComplex<Real>* sorted_M, int lds, const Triple<Real>* config1,
                            const Triple<Real>* config2) {
  const int id_i = blockIdx.x * blockDim.x + threadIdx.x;
  const int id_j = blockIdx.y * blockDim.y + threadIdx.y;
  if (id_i >= size || id_j >= size)
    return;

  const int inp_i = config1[id_i].idx;
  const int inp_j = config2[id_j].idx;

  sorted_M[id_i + lds * id_j].x = M[inp_i + ldm * inp_j];
  sorted_M[id_i + lds * id_j].y = 0;
}

template <typename InpScalar, typename Real>
void sortM(const int size, const InpScalar* M, const int ldm, std::complex<Real>* sorted_M,
           const int lds, const Triple<Real>* config1, const Triple<Real>* config2,
           const hipStream_t stream) {
  if (!size)
    return;

  auto const blocks = getBlockSize(size, size);

  sortMKernel<<<blocks[0], blocks[1], 0, stream>>>(size, M, ldm, castCudaComplex(sorted_M), lds,
                                                   config1, config2);
}

template <typename Real>
__global__ void computeTKernel(const int n, const int m, CudaComplex<Real>* T, int ldt,
                               const Triple<Real>* config, const Real* w, const bool transposed) {
  const int id_i = blockIdx.x * blockDim.x + threadIdx.x;
  const int id_j = blockIdx.y * blockDim.y + threadIdx.y;
  if (id_i >= n || id_j >= m)
    return;

  if (!transposed) {
    T[id_i + ldt * id_j].x = cos(w[id_i] * config[id_j].tau);
    T[id_i + ldt * id_j].y = sin(w[id_i] * config[id_j].tau);
  }
  else {
    T[id_i + ldt * id_j].x = cos(w[id_j] * config[id_i].tau);
    T[id_i + ldt * id_j].y = -sin(w[id_j] * config[id_i].tau);
  }
}

template <typename Real>
void computeT(const int n, const int m, std::complex<Real>* T, int ldt, const Triple<Real>* config,
              const Real* w, const bool transposed, const hipStream_t stream) {
  auto const blocks = getBlockSize(n, m);

  computeTKernel<<<blocks[0], blocks[1], 0, stream>>>(n, m, castCudaComplex(T), ldt, config, w,
                                                      transposed);
}

template <typename Real>
__global__ void rearrangeOutputKernel(const int nw, const int no, const int nb,
                                      const CudaComplex<Real>* in, const int ldi,
                                      CudaComplex<Real>* out, const int ldo) {
  const int id_i = blockIdx.x * blockDim.x + threadIdx.x;
  const int id_j = blockIdx.y * blockDim.y + threadIdx.y;
  const int n_rows = nw / 2 * no;
  const int n_cols = nw * no;
  if (id_i >= n_rows || id_j >= n_cols)
    return;

  auto get_indices = [nb](int id, const int nw, int& b, int& r, int& w) {
    r = id / (nw * nb);
    id -= r * nw * nb;
    b = id / nw;
    w = id - b * nw;
  };
  int w1, w2, b1, b2, r1, r2;

  get_indices(id_i, nw / 2, b1, r1, w1);
  get_indices(id_j, nw, b2, r2, w2);

  const int nr = no / nb;
  const int out_i = r1 + nr * b1 + no * w1;
  const int out_j = r2 + nr * b2 + no * w2;

  out[out_i + ldo * out_j] = in[id_i + ldi * id_j];
}

template <typename Real>
void rearrangeOutput(const int nw, const int no, const int nb, const std::complex<Real>* in,
                     const int ldi, std::complex<Real>* out, const int ldo,
                     const hipStream_t stream) {
  const int n_rows = nw / 2 * no;
  const int n_cols = nw * no;
  auto const blocks = getBlockSize(n_rows, n_cols);

  rearrangeOutputKernel<Real><<<blocks[0], blocks[1], 0, stream>>>(nw, no, nb, castCudaComplex(in),
                                                                   ldi, castCudaComplex(out), ldo);
}

// Explicit instantiation.
template void sortM<double, double>(int, const double*, int, std::complex<double>*, int,
                                    const Triple<double>*, const Triple<double>*, const hipStream_t);
template void sortM<double, float>(int, const double*, int, std::complex<float>*, int,
                                   const Triple<float>*, const Triple<float>*,
                                   const hipStream_t stream);
template void sortM<float, double>(int, const float*, int, std::complex<double>*, int,
                                   const Triple<double>*, const Triple<double>*,
                                   const hipStream_t stream);
template void sortM<float, float>(int, const float*, int, std::complex<float>*, int,
                                  const Triple<float>*, const Triple<float>*,
                                  const hipStream_t stream);

template void computeT<double>(int, int, std::complex<double>*, int, const Triple<double>*,
                               const double*, bool, const hipStream_t);
template void computeT<float>(int, int, std::complex<float>*, int, const Triple<float>*,
                              const float*, bool, const hipStream_t);

template void rearrangeOutput<double>(const int nw, const int no, const int nb,
                                      const std::complex<double>* in, const int ldi,
                                      std::complex<double>* out, const int ldo,
                                      const hipStream_t stream);
template void rearrangeOutput<float>(const int nw, const int no, const int nb,
                                     const std::complex<float>* in, const int ldi,
                                     std::complex<float>* out, const int ldo,
                                     const hipStream_t stream);

}  // namespace details
}  // namespace accumulator
}  // namespace solver
}  // namespace phys
}  // namespace dca
