#include "hip/hip_runtime.h"
// Copyright (C) 2018 ETH Zurich
// Copyright (C) 2018 UT-Battelle, LLC
// All rights reserved.
//
// See LICENSE.txt for terms of usage.
// See CITATION.txt for citation guidelines if you use this code for scientific publications.
//
// Author: Giovanni Balduzzi (gbalduzz@itp.phys.ethz.ch)
//         Weile Wei (wwei9@lsu.ed)
//
// Implements the GPU kernels used by the DFT algorithm.

#include "dca/phys/dca_step/cluster_solver/shared_tools/accumulation/tp/kernels_interface.hpp"

#include <array>
#include <cassert>
#include <complex>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "dca/parallel/util/get_workload.hpp"
#include "dca/util/integer_division.hpp"
#include "dca/linalg/util/cast_cuda.hpp"
#include "dca/linalg/util/atomic_add_cuda.cu.hpp"
#include "dca/linalg/util/complex_operators_cuda.cu.hpp"
#include "dca/linalg/util/error_cuda.hpp"
#include "dca/phys/dca_step/cluster_solver/shared_tools/accumulation/tp/g4_helper.cuh"
#include "dca/phys/four_point_type.hpp"

namespace dca {
namespace phys {
namespace solver {
namespace accumulator {
namespace details {
// dca::phys::solver::accumulator::details::

using namespace linalg;
using linalg::util::CudaComplex;
using linalg::util::castCudaComplex;

std::array<dim3, 2> getBlockSize(const uint i, const uint j, const uint block_size = 32) {
  const uint n_threads_i = std::min(block_size, i);
  const uint n_threads_j = std::min(block_size, j);
  if (n_threads_i * n_threads_j > 32 * 32)
    throw(std::logic_error("Block size is too big"));

  const uint n_blocks_i = dca::util::ceilDiv(i, n_threads_i);
  const uint n_blocks_j = dca::util::ceilDiv(j, n_threads_j);

  return std::array<dim3, 2>{dim3(n_blocks_i, n_blocks_j), dim3(n_threads_i, n_threads_j)};
}

template <typename Real>
__global__ void computeGSinglebandKernel(CudaComplex<Real>* __restrict__ G, int ldg,
                                         const CudaComplex<Real>* __restrict__ G0, int nk,
                                         int nw_pos, const Real beta) {
  // Computes G = -G0(w1) * M(w1, w2) * G(w2) + (w1 == w2) * beta * G0(w1).

  const int n_rows = nk * nw_pos;
  const int n_cols = n_rows * 2;
  const int id_i = blockIdx.x * blockDim.x + threadIdx.x;
  const int id_j = blockIdx.y * blockDim.y + threadIdx.y;
  if (id_i >= n_rows || id_j >= n_cols)
    return;

  auto get_indices = [=](const int id, int& k, int& w) {
    w = id / nk;
    k = id - nk * w;
  };
  int w1, w2, k1, k2;
  get_indices(id_i, k1, w1);
  get_indices(id_j, k2, w2);

  const CudaComplex<Real> G0_w1 = G0[k1 + nk * (w1 + nw_pos)];
  const CudaComplex<Real> G0_w2 = G0[k2 + nk * w2];

  G[id_i + ldg * id_j] *= -G0_w1 * G0_w2;
  if (k1 == k2 && w1 + nw_pos == w2) {
    G[id_i + ldg * id_j] += G0_w1 * beta;
  }
}

template <typename Real>
void computeGSingleband(std::complex<Real>* G, int ldg, const std::complex<Real>* G0, int nk,
                        int nw_pos, const Real beta, hipStream_t stream) {
  const int n_rows = nk * nw_pos;
  auto blocks = getBlockSize(n_rows, n_rows * 2);

  computeGSinglebandKernel<<<blocks[0], blocks[1], 0, stream>>>(
      castCudaComplex(G), ldg, castCudaComplex(G0), nk, nw_pos, beta);
}

template <typename Real>
__global__ void computeGMultibandKernel(CudaComplex<Real>* __restrict__ G, int ldg,
                                        const CudaComplex<Real>* __restrict__ G0, int ldg0, int nb,
                                        int nk, int nw_pos, Real beta) {
  // Computes G = -G0(w1) * M(w1, w2) * G(w2) + (w1 == w2) * beta * G0(w1).
  // The product is to be intended as matrix-matrix multiplication in band space.

  const int id_i = blockIdx.x * blockDim.x + threadIdx.x;
  const int id_j = blockIdx.y * blockDim.y + threadIdx.y;

  if (id_i >= nb * nk * nw_pos || id_j >= nb * nk * nw_pos * 2)
    return;

  const int no = nb * nk;
  auto get_indices = [=](int id, int& b, int& k, int& w) {
    w = id / no;
    id -= w * no;
    k = id / nb;
    b = id - k * nb;
  };
  int w1, w2, k1, k2, b1, b2;
  get_indices(id_i, b1, k1, w1);
  get_indices(id_j, b2, k2, w2);
  w1 += nw_pos;

  // Note: cuda does not support templated shared memory.
  extern __shared__ char shared_mem[];
  CudaComplex<Real>* const M_block = reinterpret_cast<CudaComplex<Real>*>(shared_mem);
  const int local_row_start = (threadIdx.y / nb) * nb;
  const int local_col_start = (threadIdx.x / nb) * nb;
  const int ldm = blockDim.y;
  CudaComplex<Real>* const M = M_block + local_row_start + ldm * local_col_start;

  CudaComplex<Real>& G_val = G[id_i + ldg * id_j];
  M[b1 + ldm * b2] = G_val;
  __syncthreads();

  const CudaComplex<Real>* const G0_w1 = G0 + nb * k1 + no * w1;
  const CudaComplex<Real>* const G0_w2 = G0 + nb * k2 + no * w2;

  G_val.x = G_val.y = 0;
  for (int j = 0; j < nb; ++j) {
    const CudaComplex<Real> G0_w2_val = G0_w2[j + ldg0 * b2];
    for (int i = 0; i < nb; ++i)
      G_val -= G0_w1[b1 + ldg0 * i] * M[i + ldm * j] * G0_w2_val;
  }

  if (G0_w1 == G0_w2)
    G_val += G0_w1[b1 + ldg0 * b2] * beta;
}

template <typename Real>
void computeGMultiband(std::complex<Real>* G, int ldg, const std::complex<Real>* G0, int ldg0,
                       int nb, int nk, int nw_pos, Real beta, hipStream_t stream) {
  const int n_rows = nb * nk * nw_pos;

  auto get_block_width = [nb] {
    if (nb > 16)
      throw(std::out_of_range("Too many bands."));
    for (int candidate = 16; candidate > 0; --candidate)
      if (!(candidate % nb))
        return candidate;
    return -1;
  };

  const int width = get_block_width();
  const auto blocks = getBlockSize(n_rows, n_rows * 2, width);

  computeGMultibandKernel<<<blocks[0], blocks[1], width * width * sizeof(std::complex<Real>), stream>>>(
      castCudaComplex(G), ldg, castCudaComplex(G0), ldg0, nb, nk, nw_pos, beta);
}

template <typename Real, FourPointType type>
__global__ void updateG4Kernel(CudaComplex<Real>* __restrict__ G4,
                               const CudaComplex<Real>* __restrict__ G_up, const int ldgu,
                               const CudaComplex<Real>* __restrict__ G_down, const int ldgd,
                               const int sign, const bool atomic, const uint64_t start,
                               const uint64_t end) {
  // TODO: reduce code duplication.
  // TODO: decrease, if possible, register pressure. E.g. a single thread computes all bands.

  const uint64_t local_g4_index =
      static_cast<uint64_t>(blockIdx.x) * static_cast<uint64_t>(blockDim.x) +
      static_cast<uint64_t>(threadIdx.x);

  const uint64_t g4_index = local_g4_index + start;

  if (g4_index >= end) {  // out of domain.
    return;
  }

  unsigned b1, b2, b3, b4, k1, k2, k_ex, w1, w2, w_ex;
  g4_helper.unrollIndex(g4_index, b1, b2, b3, b4, k1, w1, k2, w2, k_ex, w_ex);

  const unsigned nb = g4_helper.get_bands();
  const unsigned nk = g4_helper.get_cluster_size();

  CudaComplex<Real> contribution;
  const unsigned no = nk * nb;
  auto cond_conj = [](const CudaComplex<Real> a, const bool cond) { return cond ? conj(a) : a; };

  // Compute the contribution to G4. In all the products of Green's function of type Ga * Gb,
  // the dependency on the bands is implied as Ga(b1, b2) * Gb(b2, b3). Sums and differences with
  // the exchange momentum, implies the same operation is performed with the exchange frequency.
  // See tp_accumulator.hpp for more details.
  switch (type) {
    case PARTICLE_HOLE_TRANSVERSE: {
      // contribution <- -\sum_s G(k1, k2, s) * G(k2 + k_ex, k1 + k_ex, -s)
      int w1_a(w1);
      int w2_a(w2);
      int k1_a(k1);
      int k2_a(k2);
      const bool conj_a = g4_helper.extendGIndices(k1_a, k2_a, w1_a, w2_a);
      const int i_a = b1 + nb * k1_a + no * w1_a;
      const int j_a = b4 + nb * k2_a + no * w2_a;

      const CudaComplex<Real> Ga_1 = cond_conj(G_up[i_a + ldgu * j_a], conj_a);
      const CudaComplex<Real> Ga_2 = cond_conj(G_down[i_a + ldgd * j_a], conj_a);

      int w1_b(g4_helper.addWex(w2, w_ex));
      int w2_b(g4_helper.addWex(w1, w_ex));
      int k1_b = g4_helper.addKex(k2, k_ex);
      int k2_b = g4_helper.addKex(k1, k_ex);
      const bool conj_b = g4_helper.extendGIndices(k1_b, k2_b, w1_b, w2_b);
      const int i_b = b2 + nb * k1_b + no * w1_b;
      const int j_b = b3 + nb * k2_b + no * w2_b;

      const CudaComplex<Real> Gb_1 = cond_conj(G_down[i_b + ldgd * j_b], conj_b);
      const CudaComplex<Real> Gb_2 = cond_conj(G_up[i_b + ldgu * j_b], conj_b);

      contribution = -(Ga_1 * Gb_1 + Ga_2 * Gb_2);
    } break;

    // The PARTICLE_HOLE_MAGNETIC contribution is computed in two parts:
    case PARTICLE_HOLE_MAGNETIC: {
      // contribution <- -\sum_s G(k1, k2, s) * G(k2 + k_ex, k1 + k_ex, s)
      int w1_a(w1);
      int w2_a(w2);
      int k1_a(k1);
      int k2_a(k2);
      const bool conj_a = g4_helper.extendGIndices(k1_a, k2_a, w1_a, w2_a);
      const int i_a = b1 + nb * k1_a + no * w1_a;
      const int j_a = b4 + nb * k2_a + no * w2_a;
      const CudaComplex<Real> Ga_1 = cond_conj(G_up[i_a + ldgu * j_a], conj_a);
      const CudaComplex<Real> Ga_2 = cond_conj(G_down[i_a + ldgd * j_a], conj_a);

      int w1_b(g4_helper.addWex(w2, w_ex));
      int w2_b(g4_helper.addWex(w1, w_ex));
      int k1_b = g4_helper.addKex(k2, k_ex);
      int k2_b = g4_helper.addKex(k1, k_ex);
      const bool conj_b = g4_helper.extendGIndices(k1_b, k2_b, w1_b, w2_b);
      const int i_b = b2 + nb * k1_b + no * w1_b;
      const int j_b = b3 + nb * k2_b + no * w2_b;

      const CudaComplex<Real> Gb_1 = cond_conj(G_up[i_b + ldgu * j_b], conj_b);

      const CudaComplex<Real> Gb_2 = cond_conj(G_down[i_b + ldgd * j_b], conj_b);

      contribution = -(Ga_1 * Gb_1 + Ga_2 * Gb_2);
    }
      // Spin Difference Contribution
      // new scope to reuse local index variables
      {
        // contribution += (\sum_s s * G(k1, k1 + k_ex)) * (\sum_s s * G(k2 + k_ex, k2))
        int w1_a(w1);
        int w2_a(g4_helper.addWex(w1, w_ex));
        int k1_a = k1;
        int k2_a = g4_helper.addKex(k1, k_ex);
        const bool conj_a = g4_helper.extendGIndices(k1_a, k2_a, w1_a, w2_a);
        const int i_a = b1 + nb * k1_a + no * w1_a;
        const int j_a = b3 + nb * k2_a + no * w2_a;

        const CudaComplex<Real> Ga =
            cond_conj(G_up[i_a + ldgu * j_a] - G_down[i_a + ldgd * j_a], conj_a);

        int w1_b(g4_helper.addWex(w2, w_ex));
        int w2_b(w2);
        int k1_b = g4_helper.addKex(k2, k_ex);
        int k2_b = k2;
        const bool conj_b = g4_helper.extendGIndices(k1_b, k2_b, w1_b, w2_b);
        const int i_b = b2 + nb * k1_b + no * w1_b;
        const int j_b = b4 + nb * k2_b + no * w2_b;

        const CudaComplex<Real> Gb =
            cond_conj(G_up[i_b + ldgu * j_b] - G_down[i_b + ldgd * j_b], conj_b);

        contribution += (Ga * Gb);
      }
      break;

    // The PARTICLE_HOLE_CHARGE contribution is computed in two parts:
    case PARTICLE_HOLE_CHARGE: {
      // contribution <- -\sum_s G(k1, k2, s) * G(k2 + k_ex, k1 + k_ex, s)
      int w1_a(w1);
      int w2_a(w2);
      int k1_a(k1);
      int k2_a(k2);
      const bool conj_a = g4_helper.extendGIndices(k1_a, k2_a, w1_a, w2_a);
      const int i_a = b1 + nb * k1_a + no * w1_a;
      const int j_a = b4 + nb * k2_a + no * w2_a;

      const CudaComplex<Real> Ga_1 = cond_conj(G_up[i_a + ldgu * j_a], conj_a);
      const CudaComplex<Real> Ga_2 = cond_conj(G_down[i_a + ldgd * j_a], conj_a);

      int w1_b(g4_helper.addWex(w2, w_ex));
      int w2_b(g4_helper.addWex(w1, w_ex));
      int k1_b = g4_helper.addKex(k2, k_ex);
      int k2_b = g4_helper.addKex(k1, k_ex);
      const bool conj_b = g4_helper.extendGIndices(k1_b, k2_b, w1_b, w2_b);
      const int i_b = b2 + nb * k1_b + no * w1_b;
      const int j_b = b3 + nb * k2_b + no * w2_b;

      const CudaComplex<Real> Gb_1 = cond_conj(G_up[i_b + ldgu * j_b], conj_b);

      const CudaComplex<Real> Gb_2 = cond_conj(G_down[i_b + ldgd * j_b], conj_b);

      contribution = -(Ga_1 * Gb_1 + Ga_2 * Gb_2);
    }
      // Spin Difference Contribution
      // new scope to reuse local index variables
      {
        // contribution += (\sum_s G(k1, k1 + k_ex, s)) * (\sum_s G(k2 + k_ex, k2, s))
        // TODO: pull into function, index setting code is identical for Spin cases
        int w1_a(w1);
        int w2_a(g4_helper.addWex(w1, w_ex));
        int k1_a = k1;
        int k2_a = g4_helper.addKex(k1, k_ex);
        const bool conj_a = g4_helper.extendGIndices(k1_a, k2_a, w1_a, w2_a);
        const int i_a = b1 + nb * k1_a + no * w1_a;
        const int j_a = b3 + nb * k2_a + no * w2_a;

        const CudaComplex<Real> Ga =
            cond_conj(G_up[i_a + ldgu * j_a] + G_down[i_a + ldgd * j_a], conj_a);

        int w1_b(g4_helper.addWex(w2, w_ex));
        int w2_b(w2);
        int k1_b = g4_helper.addKex(k2, k_ex);
        int k2_b = k2;
        const bool conj_b = g4_helper.extendGIndices(k1_b, k2_b, w1_b, w2_b);
        const int i_b = b2 + nb * k1_b + no * w1_b;
        const int j_b = b4 + nb * k2_b + no * w2_b;

        const CudaComplex<Real> Gb =
            cond_conj(G_up[i_b + ldgu * j_b] + G_down[i_b + ldgd * j_b], conj_b);

        contribution += (Ga * Gb);
      }
      break;

      // The PARTICLE_HOLE_LONGITUDINAL_UP_UP contribution is computed in two parts:
    case PARTICLE_HOLE_LONGITUDINAL_UP_UP: {
      // contribution <- \sum_s G(k1, k1+k_ex, s) * G(k2+k_ex, k2, s)
      int w1_a(w1);
      int w2_a(g4_helper.addWex(w1, w_ex));
      int k1_a = k1;
      int k2_a = g4_helper.addKex(k1, k_ex);
      const bool conj_a = g4_helper.extendGIndices(k1_a, k2_a, w1_a, w2_a);
      const int i_a = b1 + nb * k1_a + no * w1_a;
      const int j_a = b3 + nb * k2_a + no * w2_a;

      const CudaComplex<Real> Ga_1 = cond_conj(G_up[i_a + ldgu * j_a], conj_a);
      const CudaComplex<Real> Ga_2 = cond_conj(G_down[i_a + ldgd * j_a], conj_a);

      int w1_b(g4_helper.addWex(w2, w_ex));
      int w2_b(w2);
      int k1_b = g4_helper.addKex(k2, k_ex);
      int k2_b = k2;
      const bool conj_b = g4_helper.extendGIndices(k1_b, k2_b, w1_b, w2_b);
      const int i_b = b2 + nb * k1_b + no * w1_b;
      const int j_b = b4 + nb * k2_b + no * w2_b;

      const CudaComplex<Real> Gb_1 = cond_conj(G_up[i_b + ldgd * j_b], conj_b);
      const CudaComplex<Real> Gb_2 = cond_conj(G_down[i_b + ldgu * j_b], conj_b);

      contribution = (Ga_1 * Gb_1 + Ga_2 * Gb_2);
    }
      {
        // contribution <- -\sum_s G(k1, k2, s) * G(k2 + k_ex, k1 + k_ex, s)
        int w1_a(w1);
        int w2_a(w2);
        int k1_a(k1);
        int k2_a(k2);
        const bool conj_a = g4_helper.extendGIndices(k1_a, k2_a, w1_a, w2_a);
        const int i_a = b1 + nb * k1_a + no * w1_a;
        const int j_a = b4 + nb * k2_a + no * w2_a;

        const CudaComplex<Real> Ga_1 = cond_conj(G_up[i_a + ldgu * j_a], conj_a);
        const CudaComplex<Real> Ga_2 = cond_conj(G_down[i_a + ldgd * j_a], conj_a);

        int w1_b(g4_helper.addWex(w2, w_ex));
        int w2_b(g4_helper.addWex(w1, w_ex));
        int k1_b = g4_helper.addKex(k2, k_ex);
        int k2_b = g4_helper.addKex(k1, k_ex);
        const bool conj_b = g4_helper.extendGIndices(k1_b, k2_b, w1_b, w2_b);
        const int i_b = b2 + nb * k1_b + no * w1_b;
        const int j_b = b3 + nb * k2_b + no * w2_b;

        const CudaComplex<Real> Gb_1 = cond_conj(G_up[i_b + ldgd * j_b], conj_b);
        const CudaComplex<Real> Gb_2 = cond_conj(G_down[i_b + ldgu * j_b], conj_b);

        contribution += -(Ga_1 * Gb_1 + Ga_2 * Gb_2);
      }
      break;

    case PARTICLE_HOLE_LONGITUDINAL_UP_DOWN: {
      // contribution <- \sum_s G(k1, k1+k_ex, s) * G(k2+k_ex, k2, -s)
      int w1_a(w1);
      int w2_a(g4_helper.addWex(w1, w_ex));
      int k1_a = k1;
      int k2_a = g4_helper.addKex(k1, k_ex);
      const bool conj_a = g4_helper.extendGIndices(k1_a, k2_a, w1_a, w2_a);
      const int i_a = b1 + nb * k1_a + no * w1_a;
      const int j_a = b3 + nb * k2_a + no * w2_a;

      const CudaComplex<Real> Ga_1 = cond_conj(G_up[i_a + ldgu * j_a], conj_a);
      const CudaComplex<Real> Ga_2 = cond_conj(G_down[i_a + ldgd * j_a], conj_a);

      int w1_b(g4_helper.addWex(w2, w_ex));
      int w2_b(w2);
      int k1_b = g4_helper.addKex(k2, k_ex);
      int k2_b = k2;
      const bool conj_b = g4_helper.extendGIndices(k1_b, k2_b, w1_b, w2_b);
      const int i_b = b2 + nb * k1_b + no * w1_b;
      const int j_b = b4 + nb * k2_b + no * w2_b;

      const CudaComplex<Real> Gb_1 = cond_conj(G_down[i_b + ldgd * j_b], conj_b);
      const CudaComplex<Real> Gb_2 = cond_conj(G_up[i_b + ldgu * j_b], conj_b);

      contribution = (Ga_1 * Gb_1 + Ga_2 * Gb_2);
    } break;

    case PARTICLE_PARTICLE_UP_DOWN: {
      // contribution <- -\sum_s G(k_ex - k2, k_ex - k1, s) * G(k2, k1, -s).
      int w1_a(w1);
      int w2_a(w2);
      int k1_a(k1);
      int k2_a(k2);
      const bool conj_a = g4_helper.extendGIndices(k1_a, k2_a, w1_a, w2_a);
      const int i_a = b1 + nb * k1_a + no * w1_a;
      const int j_a = b3 + nb * k2_a + no * w2_a;

      const CudaComplex<Real> Ga_1 = cond_conj(G_up[i_a + ldgu * j_a], conj_a);
      const CudaComplex<Real> Ga_2 = cond_conj(G_down[i_a + ldgd * j_a], conj_a);

      int w1_b(g4_helper.wexMinus(w1, w_ex));
      int w2_b(g4_helper.wexMinus(w2, w_ex));
      int k1_b = g4_helper.kexMinus(k1, k_ex);
      int k2_b = g4_helper.kexMinus(k2, k_ex);
      const bool conj_b = g4_helper.extendGIndices(k1_b, k2_b, w1_b, w2_b);
      const int i_b = b2 + nb * k1_b + no * w1_b;
      const int j_b = b4 + nb * k2_b + no * w2_b;

      const CudaComplex<Real> Gb_1 = cond_conj(G_down[i_b + ldgd * j_b], conj_b);
      const CudaComplex<Real> Gb_2 = cond_conj(G_up[i_b + ldgu * j_b], conj_b);

      contribution = (Ga_1 * Gb_1 + Ga_2 * Gb_2);
    } break;
    default:  // abort
      asm("trap;");
  }

  CudaComplex<Real>* const result_ptr = G4 + local_g4_index;
  if (atomic)
    dca::linalg::atomicAdd(result_ptr, contribution * 0.5 * sign);
  else
    *result_ptr += contribution * 0.5 * sign;
}

template <typename Real, FourPointType type>
float updateG4(std::complex<Real>* G4, const std::complex<Real>* G_up, const int ldgu,
               const std::complex<Real>* G_down, const int ldgd, const int sign, bool atomic,
               hipStream_t stream, std::size_t start, std::size_t end) {
  constexpr const std::size_t n_threads = 256;
  const unsigned n_blocks = dca::util::ceilDiv(end - start, n_threads);

  updateG4Kernel<Real, type>
      <<<n_blocks, n_threads, 0, stream>>>(castCudaComplex(G4), castCudaComplex(G_up), ldgu,
                                           castCudaComplex(G_down), ldgd, sign, atomic, start, end);

  // Check for errors.
  auto err = hipPeekAtLastError();
  if (err != hipSuccess) {
    linalg::util::printErrorMessage(err, __FUNCTION__, __FILE__, __LINE__);
    throw(std::runtime_error("CUDA failed to launch the G4 kernel."));
  }

  const std::size_t n_updates = end - start;
  switch (type) {
      // Note: sign flips  are ignored and a single complex * real multiplication is
      // present in all modes.
    case PARTICLE_HOLE_TRANSVERSE:
      // Each update of a G4 entry involves 2 complex additions and 2 complex multiplications.
      return 18. * n_updates;
    case PARTICLE_HOLE_MAGNETIC:
      // Each update of a G4 entry involves 3 complex additions and 3 complex multiplications.
      return 26. * n_updates;
    case PARTICLE_HOLE_CHARGE:
      // Each update of a G4 entry involves 3 complex additions and 3 complex multiplications.
      return 26. * n_updates;
    case PARTICLE_HOLE_LONGITUDINAL_UP_UP:
      // Each update of a G4 entry involves 3 complex additions and 4 complex multiplications.
      return 32 * n_updates;
    case PARTICLE_HOLE_LONGITUDINAL_UP_DOWN:
      // Each update of a G4 entry involves 2 complex additions and 2 complex multiplications.
      return 18. * n_updates;
    case PARTICLE_PARTICLE_UP_DOWN:
      // Each update of a G4 entry involves 2 complex additions and 2 complex multiplications.
      return 18. * n_updates;
    default:
      throw(std::logic_error("Invalid mode"));
  }
}

// Explicit instantiation.
template void computeGSingleband<float>(std::complex<float>* G, int ldg,
                                        const std::complex<float>* G0, int nk, int nw,
                                        const float beta, hipStream_t stream);
template void computeGMultiband<float>(std::complex<float>* G, int ldg,
                                       const std::complex<float>* G0, int ldg0, int nb, int nk,
                                       int nw, float beta, hipStream_t stream);

template void computeGSingleband<double>(std::complex<double>* G, int ldg,
                                         const std::complex<double>* G0, int nk, int nw_pos,
                                         const double beta, hipStream_t stream);
template void computeGMultiband<double>(std::complex<double>* G, int ldg,
                                        const std::complex<double>* G0, int ldg0, int nb, int nk,
                                        int nw_pos, double beta, hipStream_t stream);

template float updateG4<float, PARTICLE_HOLE_TRANSVERSE>(
    std::complex<float>* G4, const std::complex<float>* G_up, const int ldgu,
    const std::complex<float>* G_down, const int ldgd, const int sign, bool atomic,
    hipStream_t stream, std::size_t start, std::size_t end);

template float updateG4<float, PARTICLE_HOLE_MAGNETIC>(
    std::complex<float>* G4, const std::complex<float>* G_up, const int ldgu,
    const std::complex<float>* G_down, const int ldgd, const int sign, bool atomic,
    hipStream_t stream, std::size_t start, std::size_t end);

template float updateG4<float, PARTICLE_HOLE_CHARGE>(std::complex<float>* G4,
                                                     const std::complex<float>* G_up, const int ldgu,
                                                     const std::complex<float>* G_down,
                                                     const int ldgd, const int sign, bool atomic,
                                                     hipStream_t stream, std::size_t start,
                                                     std::size_t end);

template float updateG4<float, PARTICLE_HOLE_LONGITUDINAL_UP_UP>(
    std::complex<float>* G4, const std::complex<float>* G_up, const int ldgu,
    const std::complex<float>* G_down, const int ldgd, const int sign, bool atomic,
    hipStream_t stream, std::size_t start, std::size_t end);

template float updateG4<float, PARTICLE_HOLE_LONGITUDINAL_UP_DOWN>(
    std::complex<float>* G4, const std::complex<float>* G_up, const int ldgu,
    const std::complex<float>* G_down, const int ldgd, const int sign, bool atomic,
    hipStream_t stream, std::size_t start, std::size_t end);

template float updateG4<float, PARTICLE_PARTICLE_UP_DOWN>(
    std::complex<float>* G4, const std::complex<float>* G_up, const int ldgu,
    const std::complex<float>* G_down, const int ldgd, const int sign, bool atomic,
    hipStream_t stream, std::size_t start, std::size_t end);

template float updateG4<double, PARTICLE_HOLE_TRANSVERSE>(
    std::complex<double>* G4, const std::complex<double>* G_up, const int ldgu,
    const std::complex<double>* G_down, const int ldgd, const int sign, bool atomic,
    hipStream_t stream, std::size_t start, std::size_t end);

template float updateG4<double, PARTICLE_HOLE_MAGNETIC>(
    std::complex<double>* G4, const std::complex<double>* G_up, const int ldgu,
    const std::complex<double>* G_down, const int ldgd, const int sign, bool atomic,
    hipStream_t stream, std::size_t start, std::size_t end);

template float updateG4<double, PARTICLE_HOLE_CHARGE>(
    std::complex<double>* G4, const std::complex<double>* G_up, const int ldgu,
    const std::complex<double>* G_down, const int ldgd, const int sign, bool atomic,
    hipStream_t stream, std::size_t start, std::size_t end);

template float updateG4<double, PARTICLE_HOLE_LONGITUDINAL_UP_UP>(
    std::complex<double>* G4, const std::complex<double>* G_up, const int ldgu,
    const std::complex<double>* G_down, const int ldgd, const int sign, bool atomic,
    hipStream_t stream, std::size_t start, std::size_t end);

template float updateG4<double, PARTICLE_HOLE_LONGITUDINAL_UP_DOWN>(
    std::complex<double>* G4, const std::complex<double>* G_up, const int ldgu,
    const std::complex<double>* G_down, const int ldgd, const int sign, bool atomic,
    hipStream_t stream, std::size_t start, std::size_t end);

template float updateG4<double, PARTICLE_PARTICLE_UP_DOWN>(
    std::complex<double>* G4, const std::complex<double>* G_up, const int ldgu,
    const std::complex<double>* G_down, const int ldgd, const int sign, bool atomic,
    hipStream_t stream, std::size_t start, std::size_t end);

}  // namespace details
}  // namespace accumulator
}  // namespace solver
}  // namespace phys
}  // namespace dca
