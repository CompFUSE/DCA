#include "hip/hip_runtime.h"
// Copyright (C) 2018 ETH Zurich
// Copyright (C) 2018 UT-Battelle, LLC
// All rights reserved.
//
// See LICENSE.txt for terms of usage.
// See CITATION.txt for citation guidelines if you use this code for scientific publications.
//
// Author: Giovanni Balduzzi (gbalduzz@itp.phys.ethz.ch)
//
// Implements the GPU kernels used by the DFT algorithm.

#include "dca/phys/dca_step/cluster_solver/shared_tools/accumulation/tp/kernels_interface.hpp"

#include <array>
#include <cassert>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "dca/util/integer_division.hpp"
#include "dca/linalg/util/cast_cuda.hpp"
#include "dca/linalg/util/atomic_add_cuda.cu.hpp"
#include "dca/linalg/util/complex_operators_cuda.cu.hpp"
#include "dca/linalg/util/error_cuda.hpp"
#include "dca/phys/dca_step/cluster_solver/shared_tools/accumulation/tp/g4_helper.cuh"

namespace dca {
namespace phys {
namespace solver {
namespace accumulator {
namespace details {
// dca::phys::solver::accumulator::details::

using namespace linalg;
using linalg::util::CudaComplex;
using linalg::util::castCudaComplex;

std::array<dim3, 2> getBlockSize(const uint i, const uint j, const uint block_size = 32) {
  const uint n_threads_i = std::min(block_size, i);
  const uint n_threads_j = std::min(block_size, j);
  if (n_threads_i * n_threads_j > 32 * 32)
    throw(std::logic_error("Block size is too big"));

  const uint n_blocks_i = dca::util::ceilDiv(i, n_threads_i);
  const uint n_blocks_j = dca::util::ceilDiv(j, n_threads_j);

  return std::array<dim3, 2>{dim3(n_blocks_i, n_blocks_j), dim3(n_threads_i, n_threads_j)};
}

std::array<dim3, 2> getBlockSize3D(const uint i, const uint j, const uint k) {
  const uint n_threads_k = std::min(uint(8), k);
  const uint max_block_size_ij = n_threads_k > 1 ? 8 : 32;
  const uint n_threads_i = std::min(max_block_size_ij, i);
  const uint n_threads_j = std::min(max_block_size_ij, j);

  const uint n_blocks_i = dca::util::ceilDiv(i, n_threads_i);
  const uint n_blocks_j = dca::util::ceilDiv(j, n_threads_j);
  const uint n_blocks_k = dca::util::ceilDiv(k, n_threads_k);

  return std::array<dim3, 2>{dim3(n_blocks_i, n_blocks_j, n_blocks_k),
                             dim3(n_threads_i, n_threads_j, n_blocks_k)};
}

template <typename Real>
__global__ void computeGSinglebandKernel(CudaComplex<Real>* __restrict__ G, int ldg,
                                         const CudaComplex<Real>* __restrict__ G0, int nk,
                                         int nw_pos, const Real beta) {
  // Computes G = -G0(w1) * M(w1, w2) * G(w2) + (w1 == w2) * beta * G0(w1).

  const int n_rows = nk * nw_pos;
  const int n_cols = n_rows * 2;
  const int id_i = blockIdx.x * blockDim.x + threadIdx.x;
  const int id_j = blockIdx.y * blockDim.y + threadIdx.y;
  if (id_i >= n_rows || id_j >= n_cols)
    return;

  auto get_indices = [=](const int id, int& k, int& w) {
    w = id / nk;
    k = id - nk * w;
  };
  int w1, w2, k1, k2;
  get_indices(id_i, k1, w1);
  get_indices(id_j, k2, w2);

  const CudaComplex<Real> G0_w1 = G0[k1 + nk * (w1 + nw_pos)];
  const CudaComplex<Real> G0_w2 = G0[k2 + nk * w2];

  G[id_i + ldg * id_j] *= -G0_w1 * G0_w2;
  if (k1 == k2 && w1 + nw_pos == w2) {
    G[id_i + ldg * id_j] += G0_w1 * beta;
  }
}

template <typename Real>
void computeGSingleband(std::complex<Real>* G, int ldg, const std::complex<Real>* G0, int nk,
                        int nw_pos, const Real beta, hipStream_t stream) {
  const int n_rows = nk * nw_pos;
  auto blocks = getBlockSize(n_rows, n_rows * 2);

  computeGSinglebandKernel<<<blocks[0], blocks[1], 0, stream>>>(
      castCudaComplex(G), ldg, castCudaComplex(G0), nk, nw_pos, beta);
}

template <typename Real>
__global__ void computeGMultibandKernel(CudaComplex<Real>* __restrict__ G, int ldg,
                                        const CudaComplex<Real>* __restrict__ G0, int ldg0, int nb,
                                        int nk, int nw_pos, Real beta) {
  // Computes G = -G0(w1) * M(w1, w2) * G(w2) + (w1 == w2) * beta * G0(w1).
  // The product is to be intended as matrix-matrix multiplication in band space.

  const int id_i = blockIdx.x * blockDim.x + threadIdx.x;
  const int id_j = blockIdx.y * blockDim.y + threadIdx.y;

  if (id_i >= nb * nk * nw_pos || id_j >= nb * nk * nw_pos * 2)
    return;

  const int no = nb * nk;
  auto get_indices = [=](int id, int& b, int& k, int& w) {
    w = id / no;
    id -= w * no;
    k = id / nb;
    b = id - k * nb;
  };
  int w1, w2, k1, k2, b1, b2;
  get_indices(id_i, b1, k1, w1);
  get_indices(id_j, b2, k2, w2);
  w1 += nw_pos;

  // Note: cuda does not support templated shared memory.
  extern __shared__ char shared_mem[];
  CudaComplex<Real>* const M_block = reinterpret_cast<CudaComplex<Real>*>(shared_mem);
  const int local_row_start = (threadIdx.y / nb) * nb;
  const int local_col_start = (threadIdx.x / nb) * nb;
  const int ldm = blockDim.y;
  CudaComplex<Real>* const M = M_block + local_row_start + ldm * local_col_start;

  CudaComplex<Real>& G_val = G[id_i + ldg * id_j];
  M[b1 + ldm * b2] = G_val;
  __syncthreads();

  const CudaComplex<Real>* const G0_w1 = G0 + nb * k1 + no * w1;
  const CudaComplex<Real>* const G0_w2 = G0 + nb * k2 + no * w2;

  G_val.x = G_val.y = 0;
  for (int j = 0; j < nb; ++j) {
    const CudaComplex<Real> G0_w2_val = G0_w2[j + ldg0 * b2];
    for (int i = 0; i < nb; ++i)
      G_val -= G0_w1[b1 + ldg0 * i] * M[i + ldm * j] * G0_w2_val;
  }

  if (G0_w1 == G0_w2)
    G_val += G0_w1[b1 + ldg0 * b2] * beta;
}

template <typename Real>
void computeGMultiband(std::complex<Real>* G, int ldg, const std::complex<Real>* G0, int ldg0,
                       int nb, int nk, int nw_pos, Real beta, hipStream_t stream) {
  const int n_rows = nb * nk * nw_pos;

  auto get_block_width = [nb] {
    if (nb > 16)
      throw(std::out_of_range("Too many bands."));
    for (int candidate = 16; candidate > 0; --candidate)
      if (!(candidate % nb))
        return candidate;
    return -1;
  };

  const int width = get_block_width();
  const auto blocks = getBlockSize(n_rows, n_rows * 2, width);

  computeGMultibandKernel<<<blocks[0], blocks[1], width * width * sizeof(std::complex<Real>), stream>>>(
      castCudaComplex(G), ldg, castCudaComplex(G0), ldg0, nb, nk, nw_pos, beta);
}

template <typename Real, FourPointType type>
__global__ void updateG4Kernel(CudaComplex<Real>* __restrict__ G4,
                               const CudaComplex<Real>* __restrict__ G_up, const int ldgu,
                               const CudaComplex<Real>* __restrict__ G_down, const int ldgd,
                               const int nb, const int nk, const int nw, const int nw_exchange,
                               const int nk_exchange, const int sign, const G4Helper helper) {
  // TODO: reduce code duplication.
  // TODO: decrease, if possible, register pressure. E.g. a single thread computes all bands.

  const int size = nk * nw * nb * nb;
  // id_i is a linearized index of b1, b2, k1, k2.
  const int id_i = blockIdx.x * blockDim.x + threadIdx.x;
  // id_j is a linearized index of b3, b4, k2, k_ex.
  const int id_j = blockIdx.y * blockDim.y + threadIdx.y;
  // id_z is a linearized index of k_ex, w_ex.
  const int id_z = blockIdx.z * blockDim.z + threadIdx.z;
  if (id_i >= size || id_j >= size || id_z >= nw_exchange * nk_exchange)
    return;

  // Unroll id_i and id_j.
  const int step2 = nb * nb;
  const int step1 = step2 * nk;
  auto get_indices = [=](int id, int& b1, int& b2, int& k, int& w) {
    w = id / step1;
    id -= w * step1;
    k = id / step2;
    id -= k * step2;
    b2 = id / nb;
    b1 = id - nb * b2;
  };
  int w1, w2, k1, k2, b1, b2, b3, b4;
  get_indices(id_i, b1, b2, k1, w1);
  get_indices(id_j, b3, b4, k2, w2);

  // Unroll the exchange index id_z = k_ex + nk_exchange * w_ex.
  const int w_ex = id_z / nk_exchange;
  const int k_ex = id_z - w_ex * nk_exchange;

  CudaComplex<Real> contribution;
  const int no = nk * nb;
  auto cond_conj = [](const CudaComplex<Real> a, const bool cond) { return cond ? conj(a) : a; };

  // Compute the contribution to G4. In all the products of Green's function of type Ga * Gb,
  // the dependency on the bands is implied as Ga(b1, b2) * Gb(b2, b3). Sums and differences with
  // the exchange momentum, implies the same operation is performed with the exchange frequency.
  // See tp_accumulator.hpp for more details.
  switch (type) {
    case PARTICLE_HOLE_TRANSVERSE: {
      // contribution <- -\sum_s G(k1, k2, s) * G(k2 + k_ex, k1 + k_ex, -s)
      int w1_a(w1);
      int w2_a(w2);
      int k1_a(k1);
      int k2_a(k2);
      const bool conj_a = helper.extendGIndices(k1_a, k2_a, w1_a, w2_a);
      const int i_a = b1 + nb * k1_a + no * w1_a;
      const int j_a = b4 + nb * k2_a + no * w2_a;

      const CudaComplex<Real> Ga_1 = cond_conj(G_up[i_a + ldgu * j_a], conj_a);
      const CudaComplex<Real> Ga_2 = cond_conj(G_down[i_a + ldgd * j_a], conj_a);

      int w1_b(helper.addWex(w2, w_ex));
      int w2_b(helper.addWex(w1, w_ex));
      int k1_b = helper.addKex(k2, k_ex);
      int k2_b = helper.addKex(k1, k_ex);
      const bool conj_b = helper.extendGIndices(k1_b, k2_b, w1_b, w2_b);
      const int i_b = b2 + nb * k1_b + no * w1_b;
      const int j_b = b3 + nb * k2_b + no * w2_b;

      const CudaComplex<Real> Gb_1 = cond_conj(G_down[i_b + ldgd * j_b], conj_b);
      const CudaComplex<Real> Gb_2 = cond_conj(G_up[i_b + ldgu * j_b], conj_b);

      contribution = -(Ga_1 * Gb_1 + Ga_2 * Gb_2);
    } break;

    // The PARTICLE_HOLE_MAGNETIC contribution is computed in two parts:
    case PARTICLE_HOLE_MAGNETIC: {
      // contribution <- -\sum_s G(k1, k2, s) * G(k2 + k_ex, k1 + k_ex, s)
      int w1_a(w1);
      int w2_a(w2);
      int k1_a(k1);
      int k2_a(k2);
      const bool conj_a = helper.extendGIndices(k1_a, k2_a, w1_a, w2_a);
      const int i_a = b1 + nb * k1_a + no * w1_a;
      const int j_a = b4 + nb * k2_a + no * w2_a;
      const CudaComplex<Real> Ga_1 = cond_conj(G_up[i_a + ldgu * j_a], conj_a);
      const CudaComplex<Real> Ga_2 = cond_conj(G_down[i_a + ldgd * j_a], conj_a);

      int w1_b(helper.addWex(w2, w_ex));
      int w2_b(helper.addWex(w1, w_ex));
      int k1_b = helper.addKex(k2, k_ex);
      int k2_b = helper.addKex(k1, k_ex);
      const bool conj_b = helper.extendGIndices(k1_b, k2_b, w1_b, w2_b);
      const int i_b = b2 + nb * k1_b + no * w1_b;
      const int j_b = b3 + nb * k2_b + no * w2_b;

      const CudaComplex<Real> Gb_1 = cond_conj(G_up[i_b + ldgu * j_b], conj_b);

      const CudaComplex<Real> Gb_2 = cond_conj(G_down[i_b + ldgd * j_b], conj_b);

      contribution = -(Ga_1 * Gb_1 + Ga_2 * Gb_2);
    }
      // Spin Difference Contribution
      // new scope to reuse local index variables
      {
        // contribution += (\sum_s s * G(k1, k1 + k_ex)) * (\sum_s s * G(k2 + k_ex, k2))
        int w1_a(w1);
        int w2_a(helper.addWex(w1, w_ex));
        int k1_a = k1;
        int k2_a = helper.addKex(k1, k_ex);
        const bool conj_a = helper.extendGIndices(k1_a, k2_a, w1_a, w2_a);
        const int i_a = b1 + nb * k1_a + no * w1_a;
        const int j_a = b3 + nb * k2_a + no * w2_a;

        const CudaComplex<Real> Ga =
            cond_conj(G_up[i_a + ldgu * j_a] - G_down[i_a + ldgd * j_a], conj_a);

        int w1_b(helper.addWex(w2, w_ex));
        int w2_b(w2);
        int k1_b = helper.addKex(k2, k_ex);
        int k2_b = k2;
        const bool conj_b = helper.extendGIndices(k1_b, k2_b, w1_b, w2_b);
        const int i_b = b2 + nb * k1_b + no * w1_b;
        const int j_b = b4 + nb * k2_b + no * w2_b;

        const CudaComplex<Real> Gb =
            cond_conj(G_up[i_b + ldgu * j_b] - G_down[i_b + ldgd * j_b], conj_b);

        contribution += (Ga * Gb);
      }
      break;

    // The PARTICLE_HOLE_CHARGE contribution is computed in two parts:
    case PARTICLE_HOLE_CHARGE: {
      // contribution <- -\sum_s G(k1, k2, s) * G(k2 + k_ex, k1 + k_ex, s)
      int w1_a(w1);
      int w2_a(w2);
      int k1_a(k1);
      int k2_a(k2);
      const bool conj_a = helper.extendGIndices(k1_a, k2_a, w1_a, w2_a);
      const int i_a = b1 + nb * k1_a + no * w1_a;
      const int j_a = b4 + nb * k2_a + no * w2_a;

      const CudaComplex<Real> Ga_1 = cond_conj(G_up[i_a + ldgu * j_a], conj_a);
      const CudaComplex<Real> Ga_2 = cond_conj(G_down[i_a + ldgd * j_a], conj_a);

      int w1_b(helper.addWex(w2, w_ex));
      int w2_b(helper.addWex(w1, w_ex));
      int k1_b = helper.addKex(k2, k_ex);
      int k2_b = helper.addKex(k1, k_ex);
      const bool conj_b = helper.extendGIndices(k1_b, k2_b, w1_b, w2_b);
      const int i_b = b2 + nb * k1_b + no * w1_b;
      const int j_b = b3 + nb * k2_b + no * w2_b;

      const CudaComplex<Real> Gb_1 = cond_conj(G_up[i_b + ldgu * j_b], conj_b);

      const CudaComplex<Real> Gb_2 = cond_conj(G_down[i_b + ldgd * j_b], conj_b);

      contribution = -(Ga_1 * Gb_1 + Ga_2 * Gb_2);
    }
      // Spin Difference Contribution
      // new scope to reuse local index variables
      {
        // contribution += (\sum_s G(k1, k1 + k_ex, s)) * (\sum_s G(k2 + k_ex, k2, s))
        // TODO: pull into function, index setting code is identical for Spin cases
        int w1_a(w1);
        int w2_a(helper.addWex(w1, w_ex));
        int k1_a = k1;
        int k2_a = helper.addKex(k1, k_ex);
        const bool conj_a = helper.extendGIndices(k1_a, k2_a, w1_a, w2_a);
        const int i_a = b1 + nb * k1_a + no * w1_a;
        const int j_a = b3 + nb * k2_a + no * w2_a;

        const CudaComplex<Real> Ga =
            cond_conj(G_up[i_a + ldgu * j_a] + G_down[i_a + ldgd * j_a], conj_a);

        int w1_b(helper.addWex(w2, w_ex));
        int w2_b(w2);
        int k1_b = helper.addKex(k2, k_ex);
        int k2_b = k2;
        const bool conj_b = helper.extendGIndices(k1_b, k2_b, w1_b, w2_b);
        const int i_b = b2 + nb * k1_b + no * w1_b;
        const int j_b = b4 + nb * k2_b + no * w2_b;

        const CudaComplex<Real> Gb =
            cond_conj(G_up[i_b + ldgu * j_b] + G_down[i_b + ldgd * j_b], conj_b);

        contribution += (Ga * Gb);
      }
      break;

    case PARTICLE_PARTICLE_UP_DOWN: {
      // contribution <- -\sum_s G(k_ex - k2, k_ex - k1, s) * G(k2, k1, -s).
      int w1_a(w1);
      int w2_a(w2);
      int k1_a(k1);
      int k2_a(k2);
      const bool conj_a = helper.extendGIndices(k1_a, k2_a, w1_a, w2_a);
      const int i_a = b1 + nb * k1_a + no * w1_a;
      const int j_a = b3 + nb * k2_a + no * w2_a;

      const CudaComplex<Real> Ga_1 = cond_conj(G_up[i_a + ldgu * j_a], conj_a);
      const CudaComplex<Real> Ga_2 = cond_conj(G_down[i_a + ldgd * j_a], conj_a);

      int w1_b(helper.wexMinus(w1, w_ex));
      int w2_b(helper.wexMinus(w2, w_ex));
      int k1_b = helper.kexMinus(k1, k_ex);
      int k2_b = helper.kexMinus(k2, k_ex);
      const bool conj_b = helper.extendGIndices(k1_b, k2_b, w1_b, w2_b);
      const int i_b = b2 + nb * k1_b + no * w1_b;
      const int j_b = b4 + nb * k2_b + no * w2_b;

      const CudaComplex<Real> Gb_1 = cond_conj(G_down[i_b + ldgd * j_b], conj_b);
      const CudaComplex<Real> Gb_2 = cond_conj(G_up[i_b + ldgu * j_b], conj_b);

      contribution = (Ga_1 * Gb_1 + Ga_2 * Gb_2);
    } break;
    default:  // abort
      asm("trap;");
  }

  CudaComplex<Real>* const result_ptr =
      G4 + helper.g4Index(b1, b2, b3, b4, k1, k2, k_ex, w1, w2, w_ex);

  dca::linalg::atomicAdd(result_ptr, contribution * 0.5 * sign);
}

template <typename Real, FourPointType type>
void updateG4(std::complex<Real>* G4, const std::complex<Real>* G_up, const int ldgu,
              const std::complex<Real>* G_down, const int ldgd, const int nb, const int nk,
              const int nw_pos, const int nw_exchange, const int nk_exchange, const int sign,
              hipStream_t stream) {
  const int nw = 2 * nw_pos;
  const int size_12 = nw * nk * nb * nb;
  const int size_3 = nw_exchange * nk_exchange;
  const auto blocks = getBlockSize3D(size_12, size_12, size_3);

  updateG4Kernel<Real, type><<<blocks[0], blocks[1], 0, stream>>>(
      castCudaComplex(G4), castCudaComplex(G_up), ldgu, castCudaComplex(G_down), ldgd, nb, nk, nw,
      nw_exchange, nk_exchange, sign, G4HelperManager::get_instance());

  // Check for errors.
  auto err = hipPeekAtLastError();
  if (err != hipSuccess) {
    linalg::util::printErrorMessage(err, __FUNCTION__, __FILE__, __LINE__);
    throw(std::runtime_error("CUDA failed to launch the G4 kernel."));
  }
}

// Explicit instantiation.
template void computeGSingleband<float>(std::complex<float>* G, int ldg,
                                        const std::complex<float>* G0, int nk, int nw,
                                        const float beta, hipStream_t stream);
template void computeGMultiband<float>(std::complex<float>* G, int ldg,
                                       const std::complex<float>* G0, int ldg0, int nb, int nk,
                                       int nw, float beta, hipStream_t stream);

template void computeGSingleband<double>(std::complex<double>* G, int ldg,
                                         const std::complex<double>* G0, int nk, int nw_pos,
                                         const double beta, hipStream_t stream);
template void computeGMultiband<double>(std::complex<double>* G, int ldg,
                                        const std::complex<double>* G0, int ldg0, int nb, int nk,
                                        int nw_pos, double beta, hipStream_t stream);

template void updateG4<float, PARTICLE_HOLE_TRANSVERSE>(
    std::complex<float>* G4, const std::complex<float>* G_up, const int ldgu,
    const std::complex<float>* G_down, const int ldgd, const int nb, const int nk, const int nw_pos,
    const int nw_exchange, const int nk_exchange, const int sign, hipStream_t stream);
template void updateG4<float, PARTICLE_HOLE_MAGNETIC>(
    std::complex<float>* G4, const std::complex<float>* G_up, const int ldgu,
    const std::complex<float>* G_down, const int ldgd, const int nb, const int nk, const int nw_pos,
    const int nw_exchange, const int nk_exchange, const int sign, hipStream_t stream);
template void updateG4<float, PARTICLE_HOLE_CHARGE>(std::complex<float>* G4,
                                                    const std::complex<float>* G_up, const int ldgu,
                                                    const std::complex<float>* G_down, const int ldgd,
                                                    const int nb, const int nk, const int nw_pos,
                                                    const int nw_exchange, const int nk_exchange,
                                                    const int sign, hipStream_t stream);
template void updateG4<float, PARTICLE_PARTICLE_UP_DOWN>(
    std::complex<float>* G4, const std::complex<float>* G_up, const int ldgu,
    const std::complex<float>* G_down, const int ldgd, const int nb, const int nk, const int nw_pos,
    const int nw_exchange, const int nk_exchange, const int sign, hipStream_t stream);

template void updateG4<double, PARTICLE_HOLE_TRANSVERSE>(
    std::complex<double>* G4, const std::complex<double>* G_up, const int ldgu,
    const std::complex<double>* G_down, const int ldgd, const int nb, const int nk, const int nw_pos,
    const int nw_exchange, const int nk_exchange, const int sign, hipStream_t stream);
template void updateG4<double, PARTICLE_HOLE_MAGNETIC>(
    std::complex<double>* G4, const std::complex<double>* G_up, const int ldgu,
    const std::complex<double>* G_down, const int ldgd, const int nb, const int nk, const int nw_pos,
    const int nw_exchange, const int nk_exchange, const int sign, hipStream_t stream);
template void updateG4<double, PARTICLE_HOLE_CHARGE>(
    std::complex<double>* G4, const std::complex<double>* G_up, const int ldgu,
    const std::complex<double>* G_down, const int ldgd, const int nb, const int nk, const int nw_pos,
    const int nw_exchange, const int nk_exchange, const int sign, hipStream_t stream);
template void updateG4<double, PARTICLE_PARTICLE_UP_DOWN>(
    std::complex<double>* G4, const std::complex<double>* G_up, const int ldgu,
    const std::complex<double>* G_down, const int ldgd, const int nb, const int nk, const int nw_pos,
    const int nw_exchange, const int nk_exchange, const int sign, hipStream_t stream);

}  // namespace details
}  // namespace accumulator
}  // namespace solver
}  // namespace phys
}  // namespace dca
