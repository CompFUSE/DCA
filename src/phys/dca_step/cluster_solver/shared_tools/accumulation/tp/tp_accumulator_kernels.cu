#include "hip/hip_runtime.h"
// Copyright (C) 2018 ETH Zurich
// Copyright (C) 2018 UT-Battelle, LLC
// All rights reserved.
//
// See LICENSE.txt for terms of usage.
// See CITATION.txt for citation guidelines if you use this code for scientific publications.
//
// Author: Giovanni Balduzzi (gbalduzz@itp.phys.ethz.ch)
//         Weile Wei (wwei9@lsu.ed)
//
// Implements the GPU kernels used by the DFT algorithm.

#include "dca/phys/dca_step/cluster_solver/shared_tools/accumulation/tp/kernels_interface.hpp"

#include <array>
#include <cassert>
#include <complex>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "dca/parallel/util/get_workload.hpp"
#include "dca/util/integer_division.hpp"
#include "dca/linalg/util/cast_cuda.hpp"
#include "dca/linalg/util/atomic_add_cuda.cu.hpp"
#include "dca/linalg/util/complex_operators_cuda.cu.hpp"
#include "dca/linalg/util/error_cuda.hpp"
#include "dca/phys/dca_step/cluster_solver/shared_tools/accumulation/tp/g4_helper.cuh"
#include "dca/phys/four_point_type.hpp"

#include <mpi.h>

namespace dca {
namespace phys {
namespace solver {
namespace accumulator {
namespace details {
// dca::phys::solver::accumulator::details::

using namespace linalg;
using linalg::util::CudaComplex;
using linalg::util::castCudaComplex;

// This function is used when distributed G4 is enabled
// With distributed G4 enabled, each rank only computes 1/total_mpi_ranks of 1D linearized G4
// with range (start <= g4_index) && (g4_index < end).
// Here, we create a 1D thread blocks to update local G4 and each kernel thread
// only updates one index of linearized G4. There might be at most one thread block will launch
// more threads than we need (i.e. not enough G4 index to compute in last thread block)
// and those threads will exit in kernel code.
std::array<dim3, 2> getBlockSize1D(int my_rank, int mpi_size, const uint64_t& total_G4_size) {
    uint64_t start, end;
    dca::parallel::util::getComputeRange(my_rank, mpi_size, total_G4_size, start, end);
    return std::array<dim3, 2>{dca::util::ceilDiv(end - start, static_cast<uint64_t>(256)), 256};
}

std::array<dim3, 2> getBlockSize(const uint i, const uint j, const uint block_size = 32) {
  const uint n_threads_i = std::min(block_size, i);
  const uint n_threads_j = std::min(block_size, j);
  if (n_threads_i * n_threads_j > 32 * 32)
    throw(std::logic_error("Block size is too big"));

  const uint n_blocks_i = dca::util::ceilDiv(i, n_threads_i);
  const uint n_blocks_j = dca::util::ceilDiv(j, n_threads_j);

  return std::array<dim3, 2>{dim3(n_blocks_i, n_blocks_j), dim3(n_threads_i, n_threads_j)};
}

std::array<dim3, 2> getBlockSize3D(const uint i, const uint j, const uint k) {
  const uint n_threads_k = std::min(uint(8), k);
  const uint max_block_size_ij = n_threads_k > 1 ? 8 : 32;
  const uint n_threads_i = std::min(max_block_size_ij, i);
  const uint n_threads_j = std::min(max_block_size_ij, j);

  const uint n_blocks_i = dca::util::ceilDiv(i, n_threads_i);
  const uint n_blocks_j = dca::util::ceilDiv(j, n_threads_j);
  const uint n_blocks_k = dca::util::ceilDiv(k, n_threads_k);

  return std::array<dim3, 2>{dim3(n_blocks_i, n_blocks_j, n_blocks_k),
                             dim3(n_threads_i, n_threads_j, n_threads_k)};
}

template <typename Real>
__global__ void computeGSinglebandKernel(CudaComplex<Real>* __restrict__ G, int ldg,
                                         const CudaComplex<Real>* __restrict__ G0, int nk,
                                         int nw_pos, const Real beta) {
  // Computes G = -G0(w1) * M(w1, w2) * G(w2) + (w1 == w2) * beta * G0(w1).

  const int n_rows = nk * nw_pos;
  const int n_cols = n_rows * 2;
  const int id_i = blockIdx.x * blockDim.x + threadIdx.x;
  const int id_j = blockIdx.y * blockDim.y + threadIdx.y;
  if (id_i >= n_rows || id_j >= n_cols)
    return;

  auto get_indices = [=](const int id, int& k, int& w) {
    w = id / nk;
    k = id - nk * w;
  };
  int w1, w2, k1, k2;
  get_indices(id_i, k1, w1);
  get_indices(id_j, k2, w2);

  const CudaComplex<Real> G0_w1 = G0[k1 + nk * (w1 + nw_pos)];
  const CudaComplex<Real> G0_w2 = G0[k2 + nk * w2];

  G[id_i + ldg * id_j] *= -G0_w1 * G0_w2;
  if (k1 == k2 && w1 + nw_pos == w2) {
    G[id_i + ldg * id_j] += G0_w1 * beta;
  }
}

template <typename Real>
void computeGSingleband(std::complex<Real>* G, int ldg, const std::complex<Real>* G0, int nk,
                        int nw_pos, const Real beta, hipStream_t stream) {
  const int n_rows = nk * nw_pos;
  auto blocks = getBlockSize(n_rows, n_rows * 2);

  computeGSinglebandKernel<<<blocks[0], blocks[1], 0, stream>>>(
      castCudaComplex(G), ldg, castCudaComplex(G0), nk, nw_pos, beta);
}

template <typename Real>
__global__ void computeGMultibandKernel(CudaComplex<Real>* __restrict__ G, int ldg,
                                        const CudaComplex<Real>* __restrict__ G0, int ldg0, int nb,
                                        int nk, int nw_pos, Real beta) {
  // Computes G = -G0(w1) * M(w1, w2) * G(w2) + (w1 == w2) * beta * G0(w1).
  // The product is to be intended as matrix-matrix multiplication in band space.

  const int id_i = blockIdx.x * blockDim.x + threadIdx.x;
  const int id_j = blockIdx.y * blockDim.y + threadIdx.y;

  if (id_i >= nb * nk * nw_pos || id_j >= nb * nk * nw_pos * 2)
    return;

  const int no = nb * nk;
  auto get_indices = [=](int id, int& b, int& k, int& w) {
    w = id / no;
    id -= w * no;
    k = id / nb;
    b = id - k * nb;
  };
  int w1, w2, k1, k2, b1, b2;
  get_indices(id_i, b1, k1, w1);
  get_indices(id_j, b2, k2, w2);
  w1 += nw_pos;

  // Note: cuda does not support templated shared memory.
  extern __shared__ char shared_mem[];
  CudaComplex<Real>* const M_block = reinterpret_cast<CudaComplex<Real>*>(shared_mem);
  const int local_row_start = (threadIdx.y / nb) * nb;
  const int local_col_start = (threadIdx.x / nb) * nb;
  const int ldm = blockDim.y;
  CudaComplex<Real>* const M = M_block + local_row_start + ldm * local_col_start;

  CudaComplex<Real>& G_val = G[id_i + ldg * id_j];
  M[b1 + ldm * b2] = G_val;
  __syncthreads();

  const CudaComplex<Real>* const G0_w1 = G0 + nb * k1 + no * w1;
  const CudaComplex<Real>* const G0_w2 = G0 + nb * k2 + no * w2;

  G_val.x = G_val.y = 0;
  for (int j = 0; j < nb; ++j) {
    const CudaComplex<Real> G0_w2_val = G0_w2[j + ldg0 * b2];
    for (int i = 0; i < nb; ++i)
      G_val -= G0_w1[b1 + ldg0 * i] * M[i + ldm * j] * G0_w2_val;
  }

  if (G0_w1 == G0_w2)
    G_val += G0_w1[b1 + ldg0 * b2] * beta;
}

template <typename Real>
void computeGMultiband(std::complex<Real>* G, int ldg, const std::complex<Real>* G0, int ldg0,
                       int nb, int nk, int nw_pos, Real beta, hipStream_t stream) {
  const int n_rows = nb * nk * nw_pos;

  auto get_block_width = [nb] {
    if (nb > 16)
      throw(std::out_of_range("Too many bands."));
    for (int candidate = 16; candidate > 0; --candidate)
      if (!(candidate % nb))
        return candidate;
    return -1;
  };

  const int width = get_block_width();
  const auto blocks = getBlockSize(n_rows, n_rows * 2, width);

  computeGMultibandKernel<<<blocks[0], blocks[1], width * width * sizeof(std::complex<Real>), stream>>>(
      castCudaComplex(G), ldg, castCudaComplex(G0), ldg0, nb, nk, nw_pos, beta);
}

template <typename Real, FourPointType type>
__global__ void updateG4Kernel(CudaComplex<Real>* __restrict__ G4,
                               const CudaComplex<Real>* __restrict__ G_up, const int ldgu,
                               const CudaComplex<Real>* __restrict__ G_down, const int ldgd,
                               const int nb, const int nk, const int nw, const int nw_exchange,
                               const int nk_exchange, const int sign, const bool atomic,
                               const int my_rank, const int mpi_size,
                               const uint64_t total_G4_size, bool distributed_g4_enabled) {
  // TODO: reduce code duplication.
  // TODO: decrease, if possible, register pressure. E.g. a single thread computes all bands.

  const uint64_t size = static_cast<uint64_t>(nk) * static_cast<uint64_t>(nw)
                        * static_cast<uint64_t>(nb) * static_cast<uint64_t>(nb);
  // id_i is a linearized index of b1, b2, k1, k2.
  // id_j is a linearized index of b3, b4, k2, k_ex.
  // id_z is a linearized index of k_ex, w_ex.
  uint64_t id_i, id_j, id_z;
  // global 1D linearized g4 index
  uint64_t g4_index;

  if(distributed_g4_enabled)
  {
   // With distributed g4 enabled, each rank only computes 1/total_mpi_ranks of G4. Specifically, the
   // originally flattened one dimensional G4 array will be evenly divided by total number of
   // mpi ranks into different regions (range length of each region is equal up to 1). Since
   // each rank only allocates its own portion of G4, so offsetting index is needed. Each rank
   // only computes G4 elements within correct starting and ending index range, otherwise, returns.
    const uint64_t local_g4_index = static_cast<uint64_t>(blockIdx.x) * static_cast<uint64_t>(blockDim.x)
                                    + static_cast<uint64_t>(threadIdx.x);

    // get global G4 index
    uint64_t start, end;
    g4_helper.getComputeRange(my_rank, mpi_size, total_G4_size, start, end);
    g4_index = local_g4_index + start;

    // thread exits if out of range, for example, this thread block has 256 threads while
    // only first several threads should peform G4 update.
    if(g4_index < start || g4_index > end)
      return;

    // Decomposite global G4 index into 3D to reuse existing get_indices function and id_* variables
    // Image G4 matrix is N by N by M, where N is nb * nb * nk * nw and M is k_ex * w_ex.
    id_i = g4_index % size;
    id_j = (g4_index - id_i)/size % size;
    id_z = ((g4_index - id_i)/size-id_j)/size;

    // offset global G4 index to local
    g4_index = local_g4_index;
  }
  else
  {
    id_i = blockIdx.x * blockDim.x + threadIdx.x;
    id_j = blockIdx.y * blockDim.y + threadIdx.y;
    id_z = blockIdx.z * blockDim.z + threadIdx.z;
    if (id_i >= size || id_j >= size || id_z >= nw_exchange * nk_exchange)
      return;
  }

  // Unroll id_i and id_j.
  const int step2 = nb * nb;
  const int step1 = step2 * nk;
  auto get_indices = [=](uint64_t id, int& b1, int& b2, int& k, int& w) {
    w = id / step1;
    id -= w * step1;
    k = id / step2;
    id -= k * step2;
    b2 = id / nb;
    b1 = id - nb * b2;
  };
  int w1, w2, k1, k2, b1, b2, b3, b4;
  get_indices(id_i, b1, b2, k1, w1);
  get_indices(id_j, b3, b4, k2, w2);

  // Unroll the exchange index id_z = k_ex + nk_exchange * w_ex.
  const int w_ex = id_z / nk_exchange;
  const int k_ex = id_z - w_ex * nk_exchange;

  if(!distributed_g4_enabled)
      g4_index = g4_helper.g4Index(b1, b2, b3, b4, k1, w1, k2, w2, k_ex, w_ex);

  CudaComplex<Real> contribution;
  const int no = nk * nb;
  auto cond_conj = [](const CudaComplex<Real> a, const bool cond) { return cond ? conj(a) : a; };

  // Compute the contribution to G4. In all the products of Green's function of type Ga * Gb,
  // the dependency on the bands is implied as Ga(b1, b2) * Gb(b2, b3). Sums and differences with
  // the exchange momentum, implies the same operation is performed with the exchange frequency.
  // See tp_accumulator.hpp for more details.
  switch (type) {
    case PARTICLE_HOLE_TRANSVERSE: {
      // contribution <- -\sum_s G(k1, k2, s) * G(k2 + k_ex, k1 + k_ex, -s)
      int w1_a(w1);
      int w2_a(w2);
      int k1_a(k1);
      int k2_a(k2);
      const bool conj_a = g4_helper.extendGIndices(k1_a, k2_a, w1_a, w2_a);
      const int i_a = b1 + nb * k1_a + no * w1_a;
      const int j_a = b4 + nb * k2_a + no * w2_a;

      const CudaComplex<Real> Ga_1 = cond_conj(G_up[i_a + ldgu * j_a], conj_a);
      const CudaComplex<Real> Ga_2 = cond_conj(G_down[i_a + ldgd * j_a], conj_a);

      int w1_b(g4_helper.addWex(w2, w_ex));
      int w2_b(g4_helper.addWex(w1, w_ex));
      int k1_b = g4_helper.addKex(k2, k_ex);
      int k2_b = g4_helper.addKex(k1, k_ex);
      const bool conj_b = g4_helper.extendGIndices(k1_b, k2_b, w1_b, w2_b);
      const int i_b = b2 + nb * k1_b + no * w1_b;
      const int j_b = b3 + nb * k2_b + no * w2_b;

      const CudaComplex<Real> Gb_1 = cond_conj(G_down[i_b + ldgd * j_b], conj_b);
      const CudaComplex<Real> Gb_2 = cond_conj(G_up[i_b + ldgu * j_b], conj_b);

      contribution = -(Ga_1 * Gb_1 + Ga_2 * Gb_2);
    } break;

    // The PARTICLE_HOLE_MAGNETIC contribution is computed in two parts:
    case PARTICLE_HOLE_MAGNETIC: {
      // contribution <- -\sum_s G(k1, k2, s) * G(k2 + k_ex, k1 + k_ex, s)
      int w1_a(w1);
      int w2_a(w2);
      int k1_a(k1);
      int k2_a(k2);
      const bool conj_a = g4_helper.extendGIndices(k1_a, k2_a, w1_a, w2_a);
      const int i_a = b1 + nb * k1_a + no * w1_a;
      const int j_a = b4 + nb * k2_a + no * w2_a;
      const CudaComplex<Real> Ga_1 = cond_conj(G_up[i_a + ldgu * j_a], conj_a);
      const CudaComplex<Real> Ga_2 = cond_conj(G_down[i_a + ldgd * j_a], conj_a);

      int w1_b(g4_helper.addWex(w2, w_ex));
      int w2_b(g4_helper.addWex(w1, w_ex));
      int k1_b = g4_helper.addKex(k2, k_ex);
      int k2_b = g4_helper.addKex(k1, k_ex);
      const bool conj_b = g4_helper.extendGIndices(k1_b, k2_b, w1_b, w2_b);
      const int i_b = b2 + nb * k1_b + no * w1_b;
      const int j_b = b3 + nb * k2_b + no * w2_b;

      const CudaComplex<Real> Gb_1 = cond_conj(G_up[i_b + ldgu * j_b], conj_b);

      const CudaComplex<Real> Gb_2 = cond_conj(G_down[i_b + ldgd * j_b], conj_b);

      contribution = -(Ga_1 * Gb_1 + Ga_2 * Gb_2);
    }
      // Spin Difference Contribution
      // new scope to reuse local index variables
      {
        // contribution += (\sum_s s * G(k1, k1 + k_ex)) * (\sum_s s * G(k2 + k_ex, k2))
        int w1_a(w1);
        int w2_a(g4_helper.addWex(w1, w_ex));
        int k1_a = k1;
        int k2_a = g4_helper.addKex(k1, k_ex);
        const bool conj_a = g4_helper.extendGIndices(k1_a, k2_a, w1_a, w2_a);
        const int i_a = b1 + nb * k1_a + no * w1_a;
        const int j_a = b3 + nb * k2_a + no * w2_a;

        const CudaComplex<Real> Ga =
            cond_conj(G_up[i_a + ldgu * j_a] - G_down[i_a + ldgd * j_a], conj_a);

        int w1_b(g4_helper.addWex(w2, w_ex));
        int w2_b(w2);
        int k1_b = g4_helper.addKex(k2, k_ex);
        int k2_b = k2;
        const bool conj_b = g4_helper.extendGIndices(k1_b, k2_b, w1_b, w2_b);
        const int i_b = b2 + nb * k1_b + no * w1_b;
        const int j_b = b4 + nb * k2_b + no * w2_b;

        const CudaComplex<Real> Gb =
            cond_conj(G_up[i_b + ldgu * j_b] - G_down[i_b + ldgd * j_b], conj_b);

        contribution += (Ga * Gb);
      }
      break;

    // The PARTICLE_HOLE_CHARGE contribution is computed in two parts:
    case PARTICLE_HOLE_CHARGE: {
      // contribution <- -\sum_s G(k1, k2, s) * G(k2 + k_ex, k1 + k_ex, s)
      int w1_a(w1);
      int w2_a(w2);
      int k1_a(k1);
      int k2_a(k2);
      const bool conj_a = g4_helper.extendGIndices(k1_a, k2_a, w1_a, w2_a);
      const int i_a = b1 + nb * k1_a + no * w1_a;
      const int j_a = b4 + nb * k2_a + no * w2_a;

      const CudaComplex<Real> Ga_1 = cond_conj(G_up[i_a + ldgu * j_a], conj_a);
      const CudaComplex<Real> Ga_2 = cond_conj(G_down[i_a + ldgd * j_a], conj_a);

      int w1_b(g4_helper.addWex(w2, w_ex));
      int w2_b(g4_helper.addWex(w1, w_ex));
      int k1_b = g4_helper.addKex(k2, k_ex);
      int k2_b = g4_helper.addKex(k1, k_ex);
      const bool conj_b = g4_helper.extendGIndices(k1_b, k2_b, w1_b, w2_b);
      const int i_b = b2 + nb * k1_b + no * w1_b;
      const int j_b = b3 + nb * k2_b + no * w2_b;

      const CudaComplex<Real> Gb_1 = cond_conj(G_up[i_b + ldgu * j_b], conj_b);

      const CudaComplex<Real> Gb_2 = cond_conj(G_down[i_b + ldgd * j_b], conj_b);

      contribution = -(Ga_1 * Gb_1 + Ga_2 * Gb_2);
    }
      // Spin Difference Contribution
      // new scope to reuse local index variables
      {
        // contribution += (\sum_s G(k1, k1 + k_ex, s)) * (\sum_s G(k2 + k_ex, k2, s))
        // TODO: pull into function, index setting code is identical for Spin cases
        int w1_a(w1);
        int w2_a(g4_helper.addWex(w1, w_ex));
        int k1_a = k1;
        int k2_a = g4_helper.addKex(k1, k_ex);
        const bool conj_a = g4_helper.extendGIndices(k1_a, k2_a, w1_a, w2_a);
        const int i_a = b1 + nb * k1_a + no * w1_a;
        const int j_a = b3 + nb * k2_a + no * w2_a;

        const CudaComplex<Real> Ga =
            cond_conj(G_up[i_a + ldgu * j_a] + G_down[i_a + ldgd * j_a], conj_a);

        int w1_b(g4_helper.addWex(w2, w_ex));
        int w2_b(w2);
        int k1_b = g4_helper.addKex(k2, k_ex);
        int k2_b = k2;
        const bool conj_b = g4_helper.extendGIndices(k1_b, k2_b, w1_b, w2_b);
        const int i_b = b2 + nb * k1_b + no * w1_b;
        const int j_b = b4 + nb * k2_b + no * w2_b;

        const CudaComplex<Real> Gb =
            cond_conj(G_up[i_b + ldgu * j_b] + G_down[i_b + ldgd * j_b], conj_b);

        contribution += (Ga * Gb);
      }
      break;

      // The PARTICLE_HOLE_LONGITUDINAL_UP_UP contribution is computed in two parts:
    case PARTICLE_HOLE_LONGITUDINAL_UP_UP: {
      // contribution <- \sum_s G(k1, k1+k_ex, s) * G(k2+k_ex, k2, s)
      int w1_a(w1);
      int w2_a(g4_helper.addWex(w1, w_ex));
      int k1_a = k1;
      int k2_a = g4_helper.addKex(k1, k_ex);
      const bool conj_a = g4_helper.extendGIndices(k1_a, k2_a, w1_a, w2_a);
      const int i_a = b1 + nb * k1_a + no * w1_a;
      const int j_a = b3 + nb * k2_a + no * w2_a;

      const CudaComplex<Real> Ga_1 = cond_conj(G_up[i_a + ldgu * j_a], conj_a);
      const CudaComplex<Real> Ga_2 = cond_conj(G_down[i_a + ldgd * j_a], conj_a);

      int w1_b(g4_helper.addWex(w2, w_ex));
      int w2_b(w2);
      int k1_b = g4_helper.addKex(k2, k_ex);
      int k2_b = k2;
      const bool conj_b = g4_helper.extendGIndices(k1_b, k2_b, w1_b, w2_b);
      const int i_b = b2 + nb * k1_b + no * w1_b;
      const int j_b = b4 + nb * k2_b + no * w2_b;

      const CudaComplex<Real> Gb_1 = cond_conj(G_up[i_b + ldgd * j_b], conj_b);
      const CudaComplex<Real> Gb_2 = cond_conj(G_down[i_b + ldgu * j_b], conj_b);

      contribution = (Ga_1 * Gb_1 + Ga_2 * Gb_2);
    }
      {
        // contribution <- -\sum_s G(k1, k2, s) * G(k2 + k_ex, k1 + k_ex, s)
        int w1_a(w1);
        int w2_a(w2);
        int k1_a(k1);
        int k2_a(k2);
        const bool conj_a = g4_helper.extendGIndices(k1_a, k2_a, w1_a, w2_a);
        const int i_a = b1 + nb * k1_a + no * w1_a;
        const int j_a = b4 + nb * k2_a + no * w2_a;

        const CudaComplex<Real> Ga_1 = cond_conj(G_up[i_a + ldgu * j_a], conj_a);
        const CudaComplex<Real> Ga_2 = cond_conj(G_down[i_a + ldgd * j_a], conj_a);

        int w1_b(g4_helper.addWex(w2, w_ex));
        int w2_b(g4_helper.addWex(w1, w_ex));
        int k1_b = g4_helper.addKex(k2, k_ex);
        int k2_b = g4_helper.addKex(k1, k_ex);
        const bool conj_b = g4_helper.extendGIndices(k1_b, k2_b, w1_b, w2_b);
        const int i_b = b2 + nb * k1_b + no * w1_b;
        const int j_b = b3 + nb * k2_b + no * w2_b;

        const CudaComplex<Real> Gb_1 = cond_conj(G_up[i_b + ldgd * j_b], conj_b);
        const CudaComplex<Real> Gb_2 = cond_conj(G_down[i_b + ldgu * j_b], conj_b);

        contribution += -(Ga_1 * Gb_1 + Ga_2 * Gb_2);
      }
      break;

    case PARTICLE_HOLE_LONGITUDINAL_UP_DOWN: {
      // contribution <- \sum_s G(k1, k1+k_ex, s) * G(k2+k_ex, k2, -s)
      int w1_a(w1);
      int w2_a(g4_helper.addWex(w1, w_ex));
      int k1_a = k1;
      int k2_a = g4_helper.addKex(k1, k_ex);
      const bool conj_a = g4_helper.extendGIndices(k1_a, k2_a, w1_a, w2_a);
      const int i_a = b1 + nb * k1_a + no * w1_a;
      const int j_a = b3 + nb * k2_a + no * w2_a;

      const CudaComplex<Real> Ga_1 = cond_conj(G_up[i_a + ldgu * j_a], conj_a);
      const CudaComplex<Real> Ga_2 = cond_conj(G_down[i_a + ldgd * j_a], conj_a);

      int w1_b(g4_helper.addWex(w2, w_ex));
      int w2_b(w2);
      int k1_b = g4_helper.addKex(k2, k_ex);
      int k2_b = k2;
      const bool conj_b = g4_helper.extendGIndices(k1_b, k2_b, w1_b, w2_b);
      const int i_b = b2 + nb * k1_b + no * w1_b;
      const int j_b = b4 + nb * k2_b + no * w2_b;

      const CudaComplex<Real> Gb_1 = cond_conj(G_down[i_b + ldgd * j_b], conj_b);
      const CudaComplex<Real> Gb_2 = cond_conj(G_up[i_b + ldgu * j_b], conj_b);

      contribution = (Ga_1 * Gb_1 + Ga_2 * Gb_2);
    } break;

    case PARTICLE_PARTICLE_UP_DOWN: {
      // contribution <- -\sum_s G(k_ex - k2, k_ex - k1, s) * G(k2, k1, -s).
      int w1_a(w1);
      int w2_a(w2);
      int k1_a(k1);
      int k2_a(k2);
      const bool conj_a = g4_helper.extendGIndices(k1_a, k2_a, w1_a, w2_a);
      const int i_a = b1 + nb * k1_a + no * w1_a;
      const int j_a = b3 + nb * k2_a + no * w2_a;

      const CudaComplex<Real> Ga_1 = cond_conj(G_up[i_a + ldgu * j_a], conj_a);
      const CudaComplex<Real> Ga_2 = cond_conj(G_down[i_a + ldgd * j_a], conj_a);

      int w1_b(g4_helper.wexMinus(w1, w_ex));
      int w2_b(g4_helper.wexMinus(w2, w_ex));
      int k1_b = g4_helper.kexMinus(k1, k_ex);
      int k2_b = g4_helper.kexMinus(k2, k_ex);
      const bool conj_b = g4_helper.extendGIndices(k1_b, k2_b, w1_b, w2_b);
      const int i_b = b2 + nb * k1_b + no * w1_b;
      const int j_b = b4 + nb * k2_b + no * w2_b;

      const CudaComplex<Real> Gb_1 = cond_conj(G_down[i_b + ldgd * j_b], conj_b);
      const CudaComplex<Real> Gb_2 = cond_conj(G_up[i_b + ldgu * j_b], conj_b);

      contribution = (Ga_1 * Gb_1 + Ga_2 * Gb_2);
    } break;
    default:  // abort
      asm("trap;");
  }

  CudaComplex<Real>* const result_ptr = G4 + g4_index;
  if (atomic)
    dca::linalg::atomicAdd(result_ptr, contribution * 0.5 * sign);
  else
    *result_ptr += contribution * 0.5 * sign;
}

template <typename Real, FourPointType type>
float updateG4(std::complex<Real>* G4, const std::complex<Real>* G_up, const int ldgu,
               const std::complex<Real>* G_down, const int ldgd, const int nb, const int nk,
               const int nw_pos, const int nw_exchange, const int nk_exchange, const int sign,
               bool atomic, hipStream_t stream, const int my_rank, const int mpi_size,
               const uint64_t total_G4_size, const bool distributed_g4_enabled) {
  const int nw = 2 * nw_pos;
  const int size_12 = nw * nk * nb * nb;
  const int size_3 = nw_exchange * nk_exchange;

  const auto blocks = distributed_g4_enabled ? getBlockSize1D(my_rank, mpi_size, total_G4_size) :
                        getBlockSize3D(size_12, size_12, size_3);

  updateG4Kernel<Real, type><<<blocks[0], blocks[1], 0, stream>>>(
      castCudaComplex(G4), castCudaComplex(G_up), ldgu, castCudaComplex(G_down), ldgd, nb, nk, nw,
      nw_exchange, nk_exchange, sign, atomic, my_rank, mpi_size, total_G4_size, distributed_g4_enabled);

  // Check for errors.
  auto err = hipPeekAtLastError();
  if (err != hipSuccess) {
    linalg::util::printErrorMessage(err, __FUNCTION__, __FILE__, __LINE__);
    throw(std::runtime_error("CUDA failed to launch the G4 kernel."));
  }

  const std::size_t n_updates = size_12 * size_12 * size_3;
  switch (type) {
      // Note: sign flips  are ignored and a single complex * real multiplication is
      // present in all modes.
    case PARTICLE_HOLE_TRANSVERSE:
      // Each update of a G4 entry involves 2 complex additions and 2 complex multiplications.
      return 18. * n_updates;
    case PARTICLE_HOLE_MAGNETIC:
      // Each update of a G4 entry involves 3 complex additions and 3 complex multiplications.
      return 26. * n_updates;
    case PARTICLE_HOLE_CHARGE:
      // Each update of a G4 entry involves 3 complex additions and 3 complex multiplications.
      return 26. * n_updates;
    case PARTICLE_HOLE_LONGITUDINAL_UP_UP:
      // Each update of a G4 entry involves 3 complex additions and 4 complex multiplications.
      return 32 * n_updates;
    case PARTICLE_HOLE_LONGITUDINAL_UP_DOWN:
      // Each update of a G4 entry involves 2 complex additions and 2 complex multiplications.
      return 18. * n_updates;
    case PARTICLE_PARTICLE_UP_DOWN:
      // Each update of a G4 entry involves 2 complex additions and 2 complex multiplications.
      return 18. * n_updates;
    default:
      throw(std::logic_error("Invalid mode"));
  }
}

// Explicit instantiation.
template void computeGSingleband<float>(std::complex<float>* G, int ldg,
                                        const std::complex<float>* G0, int nk, int nw,
                                        const float beta, hipStream_t stream);
template void computeGMultiband<float>(std::complex<float>* G, int ldg,
                                       const std::complex<float>* G0, int ldg0, int nb, int nk,
                                       int nw, float beta, hipStream_t stream);

template void computeGSingleband<double>(std::complex<double>* G, int ldg,
                                         const std::complex<double>* G0, int nk, int nw_pos,
                                         const double beta, hipStream_t stream);
template void computeGMultiband<double>(std::complex<double>* G, int ldg,
                                        const std::complex<double>* G0, int ldg0, int nb, int nk,
                                        int nw_pos, double beta, hipStream_t stream);

template float updateG4<float, PARTICLE_HOLE_TRANSVERSE>(
    std::complex<float>* G4, const std::complex<float>* G_up, const int ldgu,
    const std::complex<float>* G_down, const int ldgd, const int nb, const int nk, const int nw_pos,
    const int nw_exchange, const int nk_exchange, const int sign, bool atomic, hipStream_t stream,
    const int my_rank, const int mpi_size, const uint64_t total_G4_size, const bool distributed_g4_enabled);

template float updateG4<float, PARTICLE_HOLE_MAGNETIC>(
    std::complex<float>* G4, const std::complex<float>* G_up, const int ldgu,
    const std::complex<float>* G_down, const int ldgd, const int nb, const int nk, const int nw_pos,
    const int nw_exchange, const int nk_exchange, const int sign, bool atomic, hipStream_t stream,
    const int my_rank, const int mpi_size, const uint64_t total_G4_size, const bool distributed_g4_enabled);

template float updateG4<float, PARTICLE_HOLE_CHARGE>(
    std::complex<float>* G4, const std::complex<float>* G_up, const int ldgu,
    const std::complex<float>* G_down, const int ldgd, const int nb, const int nk, const int nw_pos,
    const int nw_exchange, const int nk_exchange, const int sign, bool atomic, hipStream_t stream,
    const int my_rank, const int mpi_size, const uint64_t total_G4_size, const bool distributed_g4_enabled);

template float updateG4<float, PARTICLE_HOLE_LONGITUDINAL_UP_UP>(
    std::complex<float>* G4, const std::complex<float>* G_up, const int ldgu,
    const std::complex<float>* G_down, const int ldgd, const int nb, const int nk, const int nw_pos,
    const int nw_exchange, const int nk_exchange, const int sign, bool atomic, hipStream_t stream,
    const int my_rank, const int mpi_size, const uint64_t total_G4_size, const bool distributed_g4_enabled);

template float updateG4<float, PARTICLE_HOLE_LONGITUDINAL_UP_DOWN>(
    std::complex<float>* G4, const std::complex<float>* G_up, const int ldgu,
    const std::complex<float>* G_down, const int ldgd, const int nb, const int nk, const int nw_pos,
    const int nw_exchange, const int nk_exchange, const int sign, bool atomic, hipStream_t stream,
    const int my_rank, const int mpi_size, const uint64_t total_G4_size, const bool distributed_g4_enabled);

template float updateG4<float, PARTICLE_PARTICLE_UP_DOWN>(
    std::complex<float>* G4, const std::complex<float>* G_up, const int ldgu,
    const std::complex<float>* G_down, const int ldgd, const int nb, const int nk, const int nw_pos,
    const int nw_exchange, const int nk_exchange, const int sign, bool atomic, hipStream_t stream,
    const int my_rank, const int mpi_size, const uint64_t total_G4_size, const bool distributed_g4_enabled);

template float updateG4<double, PARTICLE_HOLE_TRANSVERSE>(
    std::complex<double>* G4, const std::complex<double>* G_up, const int ldgu,
    const std::complex<double>* G_down, const int ldgd, const int nb, const int nk, const int nw_pos,
    const int nw_exchange, const int nk_exchange, const int sign, bool atomic, hipStream_t stream,
    const int my_rank, const int mpi_size, const uint64_t total_G4_size, const bool distributed_g4_enabled);

template float updateG4<double, PARTICLE_HOLE_MAGNETIC>(
    std::complex<double>* G4, const std::complex<double>* G_up, const int ldgu,
    const std::complex<double>* G_down, const int ldgd, const int nb, const int nk, const int nw_pos,
    const int nw_exchange, const int nk_exchange, const int sign, bool atomic, hipStream_t stream,
    const int my_rank, const int mpi_size, const uint64_t total_G4_size, const bool distributed_g4_enabled);

template float updateG4<double, PARTICLE_HOLE_CHARGE>(
    std::complex<double>* G4, const std::complex<double>* G_up, const int ldgu,
    const std::complex<double>* G_down, const int ldgd, const int nb, const int nk, const int nw_pos,
    const int nw_exchange, const int nk_exchange, const int sign, bool atomic, hipStream_t stream,
    const int my_rank, const int mpi_size, const uint64_t total_G4_size, const bool distributed_g4_enabled);

template float updateG4<double, PARTICLE_HOLE_LONGITUDINAL_UP_UP>(
    std::complex<double>* G4, const std::complex<double>* G_up, const int ldgu,
    const std::complex<double>* G_down, const int ldgd, const int nb, const int nk, const int nw_pos,
    const int nw_exchange, const int nk_exchange, const int sign, bool atomic, hipStream_t stream,
    const int my_rank, const int mpi_size, const uint64_t total_G4_size, const bool distributed_g4_enabled);

template float updateG4<double, PARTICLE_HOLE_LONGITUDINAL_UP_DOWN>(
    std::complex<double>* G4, const std::complex<double>* G_up, const int ldgu,
    const std::complex<double>* G_down, const int ldgd, const int nb, const int nk, const int nw_pos,
    const int nw_exchange, const int nk_exchange, const int sign, bool atomic, hipStream_t stream,
    const int my_rank, const int mpi_size, const uint64_t total_G4_size, const bool distributed_g4_enabled);

template float updateG4<double, PARTICLE_PARTICLE_UP_DOWN>(
    std::complex<double>* G4, const std::complex<double>* G_up, const int ldgu,
    const std::complex<double>* G_down, const int ldgd, const int nb, const int nk, const int nw_pos,
    const int nw_exchange, const int nk_exchange, const int sign, bool atomic, hipStream_t stream,
    const int my_rank, const int mpi_size, const uint64_t total_G4_size, const bool distributed_g4_enabled);

}  // namespace details
}  // namespace accumulator
}  // namespace solver
}  // namespace phys
}  // namespace dca
