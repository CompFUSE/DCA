#include "hip/hip_runtime.h"
// Copyright (C) 2018 ETH Zurich
// Copyright (C) 2018 UT-Battelle, LLC
// All rights reserved.
//
// See LICENSE.txt for terms of usage.
// See CITATION.txt for citation guidelines if you use this code for scientific publications.
//
// Author: Giovanni Balduzzi (gbalduzz@itp.phys.ethz.ch)
//         Weile Wei (wwei9@lsu.ed)
//         Peter Doak (doakpw@ornl.gov)
// Implements the GPU kernels used by the tp_accumulator_gpu

#include "dca/phys/dca_step/cluster_solver/shared_tools/accumulation/tp/kernels_interface.hpp"

#include <array>
#include <cassert>
#include <complex>

#include "dca/platform/dca_gpu.h"

#include "dca/parallel/util/get_workload.hpp"
#include "dca/util/integer_division.hpp"
#include "dca/util/type_help.hpp"
#include "dca/util/type_utils.hpp"
#include "dca/linalg/util/atomic_add_cuda.cu.hpp"
#include "dca/linalg/util/complex_operators_cuda.cu.hpp"
#include "dca/phys/dca_step/cluster_solver/shared_tools/accumulation/tp/g4_helper.cuh"
#include "dca/phys/four_point_type.hpp"

namespace dca {
namespace phys {
namespace solver {
namespace accumulator {
namespace details {
// dca::phys::solver::accumulator::details::

using namespace linalg;
using dca::util::ComplexAlias;
using dca::util::castGPUType;
using dca::util::RealAlias;
using phys::FourPointType;
using dca::util::SignType;

std::array<dim3, 2> getBlockSize(const uint i, const uint j, const uint block_size = 32) {
  const uint n_threads_i = std::min(block_size, i);
  const uint n_threads_j = std::min(block_size, j);
  if (n_threads_i * n_threads_j > 32 * 32)
    throw(std::logic_error("Block size is too big"));

  const uint n_blocks_i = dca::util::ceilDiv(i, n_threads_i);
  const uint n_blocks_j = dca::util::ceilDiv(j, n_threads_j);

  return std::array<dim3, 2>{dim3(n_blocks_i, n_blocks_j), dim3(n_threads_i, n_threads_j)};
}

template <typename Real>
__global__ void computeGSinglebandKernel(CudaComplex<Real>* __restrict__ G, int ldg,
                                         const CudaComplex<Real>* __restrict__ G0, int nk,
                                         int nw_freq, const Real beta, int spin) {
  // Computes G = -G0(w1) * M(w1, w2) * G(w2) + (w1 == w2) * beta * G0(w1).

  const int n_rows = nk * nw_freq;
  const int n_cols = n_rows;
  const int id_i = blockIdx.x * blockDim.x + threadIdx.x;
  const int id_j = blockIdx.y * blockDim.y + threadIdx.y;
  if (id_i >= n_rows || id_j >= n_cols)
    return;

  auto get_indices = [=](const int id, int& k, int& w) {
    w = id / nk;
    k = id - nk * w;
  };

  int w1, w2, k1, k2;
  get_indices(id_i, k1, w1);
  get_indices(id_j, k2, w2);

  const CudaComplex<Real> G0_w1 = G0[k1 + nk * w1];
  const CudaComplex<Real> G0_w2 = G0[k2 + nk * w2];

  const CudaComplex<Real> M_val  = G[id_i + ldg * id_j];
  
  G[id_i + ldg * id_j] = -G0_w1 * M_val * G0_w2;
  if (k1 == k2 && w1 == w2) {
    G[id_i + ldg * id_j] += G0_w1 * beta;
  }

  printf("%f %f %f %f %f %f -- %d %d %d %d %f,%f\n", M_val, G0_w1, G0_w2, spin, k1, k2, w1, w2, G[id_i + ldg * id_j].x, G[id_i + ldg * id_j].y);
}

template <typename Real>
void computeGSingleband(std::complex<Real>* G, int ldg, const std::complex<Real>* G0, int nk,
                        int nw_freq, const Real beta, hipStream_t stream, int spin) {
  const int n_rows = nk * nw_freq;
  auto blocks = getBlockSize(n_rows, n_rows);

  computeGSinglebandKernel<<<blocks[0], blocks[1], 0, stream>>>(castGPUType(G), ldg,
                                                                castGPUType(G0), nk, nw_freq, beta, spin);
}

template <typename Real>
__global__ void computeGMultibandKernel(CudaComplex<Real>* __restrict__ G, int ldg,
                                        const CudaComplex<Real>* __restrict__ G0, int ldg0, int nb,
                                        int nk, int nw, Real beta) {
  // Computes G = -G0(w1) * M(w1, w2) * G(w2) + (w1 == w2) * beta * G0(w1).
  // The product is to be intended as matrix-matrix multiplication in band space.

  const int id_i = blockIdx.x * blockDim.x + threadIdx.x;
  const int id_j = blockIdx.y * blockDim.y + threadIdx.y;

  if (id_i >= nb * nk * nw || id_j >= nb * nk * nw)
    return;

  const int no = nb * nk;
  auto get_indices = [=](int id, int& b, int& k, int& w) {
    w = id / no;
    id -= w * no;
    k = id / nb;
    b = id - k * nb;
  };
  int w1, w2, k1, k2, b1, b2;
  get_indices(id_i, b1, k1, w1);
  get_indices(id_j, b2, k2, w2);

  // hmmm... in CPU we now just run over the entire extended range for w1 and w2
  // w1 += nw_pos;

  // Note: cuda does not support templated shared memory.
  extern __shared__ char shared_mem[];
  CudaComplex<Real>* const M_block = reinterpret_cast<CudaComplex<Real>*>(shared_mem);
  const int local_row_start = (threadIdx.y / nb) * nb;
  const int local_col_start = (threadIdx.x / nb) * nb;
  const int ldm = blockDim.y;
  CudaComplex<Real>* const M = M_block + local_row_start + ldm * local_col_start;

  CudaComplex<Real>& G_val = G[id_i + ldg * id_j];
  M[b1 + ldm * b2] = G_val;
  __syncthreads();

  const CudaComplex<Real>* const G0_w1 = G0 + nb * k1 + no * w1;
  const CudaComplex<Real>* const G0_w2 = G0 + nb * k2 + no * w2;

  G_val.x = G_val.y = 0;
  for (int j = 0; j < nb; ++j) {
    const CudaComplex<Real> G0_w2_val = G0_w2[j + ldg0 * b2];
    for (int i = 0; i < nb; ++i) {
      const CudaComplex<Real> G_band = G0_w1[b1 + ldg0 * i] * M[i + ldm * j] * G0_w2_val;
      G_val -= G_band;
    }
  }

  if (G0_w1 == G0_w2)
    G_val += G0_w1[b1 + ldg0 * b2] * beta;

  printf("%f %f %f %f %f %f -- %d %d %d %d %d %d %f,%f\n", M[b1 + ldm * b2], G0_w1[b1 + ldg0 * b2], G0_w2[b1 + ldg0 * b2], b1, b2, k1, k2, w1, w2, G_val.x, G_val.y);
}

template <typename Real>
void computeGMultiband(std::complex<Real>* G, int ldg, const std::complex<Real>* G0, int ldg0,
                       int nb, int nk, int nw, Real beta, hipStream_t stream) {
  const int n_rows = nb * nk * nw;

  auto get_block_width = [nb] {
    if (nb > 16)
      throw(std::out_of_range("Too many bands."));
    for (int candidate = 16; candidate > 0; --candidate)
      if (!(candidate % nb))
        return candidate;
    return -1;
  };

  const int width = get_block_width();
  // what is the magic 2 for?
  const auto blocks = getBlockSize(n_rows, n_rows, width);

  computeGMultibandKernel<<<blocks[0], blocks[1], width * width * sizeof(std::complex<Real>), stream>>>(
      castGPUType(G), ldg, castGPUType(G0), ldg0, nb, nk, nw, beta);
}

// template <typename Complex>
// __device__ Complex getG(const Complex* __restrict__ G, const int ldg, int k1, int k2, int w1,
//                         int w2, const int b1, const int b2) {
//   const bool is_conj = g4_helper.extendGIndices(k1, k2, w1, w2);

//   const unsigned nb = g4_helper.get_bands();
//   const unsigned nk = g4_helper.get_cluster_size();
//   const unsigned no = nb * nk;

//   unsigned i_idx = b1 + nb * k1 + no * w1;
//   unsigned j_idx = b2 + nb * k2 + no * w2;

//   auto val = G[i_idx + ldg * j_idx];

//   if (!is_conj)
//     return val;
//   else {
//     // For Moire model G_up(-k, -wn) = conj(G_dn(k, wn))
//     // but for a given configuration, SU(2) symmetry is broken by the auxialary spin field
//     // This mean the following code is incorrect and we must extend the calculation to all
//     // frequencies w1, w2, including negative w1.
//     i_idx = 1 - b2 + nb * k1 + no * w1;
//     j_idx = 1 - b1 + nb * k2 + no * w2;
//     val = conj(G[i_idx + ldg * j_idx]);
//     // if (b1==b2)
//     //     return conj(val);
//     // else {
//     //     i_idx = b2 + nb * k1 + no * w1;
//     //     j_idx = b1 + nb * k2 + no * w2;
//     //     val = -conj(G[i_idx + ldg * j_idx]);
//     return val;
//     // }
//   }
//   // return is_conj ? conj(val) : val;
// }

template <typename Scalar, FourPointType type, typename SignType>
__global__ void updateG4Kernel(CudaComplex<RealAlias<Scalar>>* __restrict__ G4,
                               const CudaComplex<RealAlias<Scalar>>* __restrict__ G_up,
                               const int ldgu,
                               const CudaComplex<RealAlias<Scalar>>* __restrict__ G_down,
                               const int ldgd, const SignType factor, const bool atomic,
                               const uint64_t start, const uint64_t end) {
  // TODO: reduce code duplication.
  // TODO: decrease, if possible, register pressure. E.g. a single thread computes all bands.

  const uint64_t local_g4_index =
      static_cast<uint64_t>(blockIdx.x) * static_cast<uint64_t>(blockDim.x) +
      static_cast<uint64_t>(threadIdx.x);

  const uint64_t g4_index = local_g4_index + start;

  if (g4_index >= end) {  // out of domain.
    return;
  }

  Scalar complex_factor;
  dca::linalg::assign(complex_factor, factor);
  const Scalar sign_over_2 = 0.5 * complex_factor;

  int b1, b2, b3, b4, k1, k2, k_ex, w1, w2, w_ex;
  g4_helper.unrollIndex(g4_index, b1, b2, b3, b4, k1, w1, k2, w2, k_ex, w_ex);

  const int nb = g4_helper.get_bands();
  const int nk = g4_helper.get_cluster_size();

  CudaComplex<RealAlias<Scalar>> contribution;
  const unsigned no = nk * nb;
  auto cond_conj = [](const CudaComplex<RealAlias<Scalar>> a, const bool cond) {
    return cond ? conj(a) : a;
  };

  // This code needs to be repeated over and over.  This happens in getGMultiband in the cpu
  // implementation. The gpu code is structed differently so without signficant restructing this
  // can't happen in the extendGIndiciesMultiBand routines.
  auto condSwapAdd = [](int& ia, int& ib, const int ba, const int bb, const bool cond) {
    if (cond) {
      ia += bb;
      ib += ba;
    }
    else {
      ia += ba;
      ib += bb;
    }
  };
  // Compute the contribution to G4. In all the products of Green's function of type Ga * Gb,
  // the dependency on the bands is implied as Ga(b1, b2) * Gb(b2, b3). Sums and differences with
  // the exchange momentum, implies the same operation is performed with the exchange frequency.
  // See tp_accumulator.hpp for more details.
  if constexpr (type == FourPointType::PARTICLE_HOLE_TRANSVERSE) {
    // contribution <- -\sum_s G(k1, k2, s) * G(k2 + k_ex, k1 + k_ex, -s)
    int w1_a(w1);
    int w2_a(w2);
    int k1_a(k1);
    int k2_a(k2);
    const bool conj_a = g4_helper.extendGIndices(k1_a, k2_a, w1_a, w2_a);
    int i_a = nb * k1_a + no * w1_a;
    int j_a = nb * k2_a + no * w2_a;
    condSwapAdd(i_a, j_a, b1, b4, conj_a);
    const CudaComplex<RealAlias<Scalar>> Ga_1 = cond_conj(G_up[i_a + ldgu * j_a], conj_a);
    const CudaComplex<RealAlias<Scalar>> Ga_2 = cond_conj(G_down[i_a + ldgd * j_a], conj_a);

    int w1_b(g4_helper.addWex(w2, w_ex));
    int w2_b(g4_helper.addWex(w1, w_ex));
    int k1_b = g4_helper.addKex(k2, k_ex);
    int k2_b = g4_helper.addKex(k1, k_ex);
    const bool conj_b = g4_helper.extendGIndices(k1_b, k2_b, w1_b, w2_b);
    int i_b = nb * k1_b + no * w1_b;
    int j_b = nb * k2_b + no * w2_b;
    condSwapAdd(i_b, j_b, b2, b3, conj_b);
    const CudaComplex<RealAlias<Scalar>> Gb_1 = cond_conj(G_down[i_b + ldgd * j_b], conj_b);
    const CudaComplex<RealAlias<Scalar>> Gb_2 = cond_conj(G_up[i_b + ldgu * j_b], conj_b);

    contribution = -sign_over_2 * (Ga_1 * Gb_1 + Ga_2 * Gb_2);
  }
  else if constexpr (type == FourPointType::PARTICLE_HOLE_MAGNETIC) {
    // The PARTICLE_HOLE_MAGNETIC contribution is computed in two parts:
    // Spin Difference Contribution
    // new scope to reuse local index variables

    // contribution += (\sum_s s * G(k1, k1 + k_ex)) * (\sum_s s * G(k2 + k_ex, k2))
    int k1_a = k1;
    int k2_a = g4_helper.addKex(k1, k_ex);
    int k1_b = g4_helper.addKex(k2, k_ex);
    int k2_b = k2;

    int w1_a(w1);
    int w2_a(g4_helper.addWex(w2, w_ex));
    int w1_b(g4_helper.addWex(w1, w_ex));
    int w2_b(w2);

    // conj_a in this case just tells us whether to swap the band axes additions or not
    bool conj_a = false;
    // if (g4_helper.get_bands() == 1)
    //   conj_a = g4_helper.extendGIndices(k1_a, k2_a, w1_a, w2_a);
    // else
    //   conj_a = g4_helper.extendGIndicesMultiBand(k1_a, k2_a, w1_a, w2_a);
    int i_a = nb * k1_a + no * w1_a;
    int j_a = nb * k2_a + no * w2_a;
    // condSwapAdd(i_a, j_a, b1, b3, conj_a);
    // CudaComplex<RealAlias<Scalar>> Ga =
    //   G_up[i_a + ldgu * j_a] - G_down[i_a + ldgd * j_a];
    // if (i_a == j_a)
    //   Ga += (G_up[i_a + ldgu * j_a] - G_down[i_a + ldgd * j_a]) *

    bool conj_b = false;
    // if (g4_helper.get_bands() == 1)
    //   conj_b = g4_helper.extendGIndices(k1_b, k2_b, w1_b, w2_b);
    // else
    //   conj_b = g4_helper.extendGIndicesMultiBand(k1_b, k2_b, w1_b, w2_b);
    int i_b = nb * k1_b + no * w1_b;
    int j_b = nb * k2_b + no * w2_b;
    // condSwapAdd(i_b, j_b, b2, b4, conj_b);
    // CudaComplex<RealAlias<Scalar>> Gb =
    //   G_up[i_b + ldgu * j_b] - G_down[i_b + ldgd * j_b];

    // contribution = sign_over_2 * (Ga * Gb);

    // direct contribution <- -\sum_s G(k1, k2, s) * G(k2 + k_ex, k1 + k_ex, s)
    k1_a = k1;
    k2_a = k2;
    k1_b = g4_helper.addKex(k2, k_ex);
    k2_b = g4_helper.addKex(k1, k_ex);

    w1_a = w1;
    w2_a = w2;
    w1_b = g4_helper.addWex(w2, w_ex);
    w2_b = g4_helper.addWex(w1, w_ex);

    if (g4_helper.get_bands() == 1)
      conj_a = g4_helper.extendGIndices(k1_a, k2_a, w1_a, w2_a);
    else
      conj_a = g4_helper.extendGIndicesMultiBand(k1_a, k2_a, w1_a, w2_a);
    i_a = nb * k1_a + no * w1_a;
    j_a = nb * k2_a + no * w2_a;
    i_a += b1;
    j_a += b3;

    CudaComplex<RealAlias<Scalar>> Ga_1 = G_up[i_a + ldgu * j_a];
    CudaComplex<RealAlias<Scalar>> Ga_2 = G_down[i_a + ldgd * j_a];

    if (g4_helper.get_bands() == 1)
      conj_b = g4_helper.extendGIndices(k1_b, k2_b, w1_b, w2_b);
    else
      conj_b = g4_helper.extendGIndicesMultiBand(k1_b, k2_b, w1_b, w2_b);

    i_b = nb * k1_b + no * w1_b;
    j_b = nb * k2_b + no * w2_b;
    i_b += b4;
    j_b += b2;
    CudaComplex<RealAlias<Scalar>> Gb_1 = G_up[i_b + ldgu * j_b];
    CudaComplex<RealAlias<Scalar>> Gb_2 = G_down[i_b + ldgd * j_b];

    contribution = -sign_over_2 * Ga_1 * Gb_1 - sign_over_2 * Ga_2 * Gb_2;
  }
  else if constexpr (type == FourPointType::PARTICLE_HOLE_CHARGE) {
    // The PARTICLE_HOLE_CHARGE contribution is computed in two parts:
    {
      // contribution <- -\sum_s G(k1, k2, s) * G(k2 + k_ex, k1 + k_ex, s)
      int w1_a(w1);
      int w2_a(w2);
      int k1_a(k1);
      int k2_a(k2);
      bool conj_a = false;
      if (g4_helper.get_bands() == 1)
        conj_a = g4_helper.extendGIndices(k1_a, k2_a, w1_a, w2_a);
      else
        conj_a = g4_helper.extendGIndicesMultiBand(k1_a, k2_a, w1_a, w2_a);
      int i_a = nb * k1_a + no * w1_a;
      int j_a = nb * k2_a + no * w2_a;
      if (conj_a) {
        i_a += b4;
        j_a += b1;
      }
      else {
        i_a += b1;
        j_a += b4;
      }

      const CudaComplex<RealAlias<Scalar>> Ga_1 = cond_conj(G_up[i_a + ldgu * j_a], conj_a);
      const CudaComplex<RealAlias<Scalar>> Ga_2 = cond_conj(G_down[i_a + ldgd * j_a], conj_a);

      int w1_b(g4_helper.addWex(w2, w_ex));
      int w2_b(g4_helper.addWex(w1, w_ex));
      int k1_b = g4_helper.addKex(k2, k_ex);
      int k2_b = g4_helper.addKex(k1, k_ex);
      bool conj_b = false;
      if (g4_helper.get_bands() == 1)
        conj_b = g4_helper.extendGIndices(k1_b, k2_b, w1_b, w2_b);
      else
        conj_b = g4_helper.extendGIndicesMultiBand(k1_b, k2_b, w1_b, w2_b);

      int i_b = nb * k1_b + no * w1_b;
      int j_b = nb * k2_b + no * w2_b;
      if (conj_b) {
        i_b += b3;
        j_b += b2;
      }
      else {
        i_b += b2;
        j_b += b3;
      }

      const CudaComplex<RealAlias<Scalar>> Gb_1 = cond_conj(G_up[i_b + ldgu * j_b], conj_b);

      const CudaComplex<RealAlias<Scalar>> Gb_2 = cond_conj(G_down[i_b + ldgd * j_b], conj_b);

      contribution = -sign_over_2 * (Ga_1 * Gb_1 + Ga_2 * Gb_2);
    }
    // Spin Difference Contribution
    // new scope to reuse local index variables
    {
      // contribution += (\sum_s G(k1, k1 + k_ex, s)) * (\sum_s G(k2 + k_ex, k2, s))
      // TODO: pull into function, index setting code is identical for Spin cases
      int w1_a(w1);
      int w2_a(g4_helper.addWex(w1, w_ex));
      int k1_a = k1;
      int k2_a = g4_helper.addKex(k1, k_ex);
      bool conj_a = false;
      if (g4_helper.get_bands() == 1)
        conj_a = g4_helper.extendGIndices(k1_a, k2_a, w1_a, w2_a);
      else
        conj_a = g4_helper.extendGIndicesMultiBand(k1_a, k2_a, w1_a, w2_a);

      int i_a = nb * k1_a + no * w1_a;
      int j_a = nb * k2_a + no * w2_a;
      if (conj_a) {
        i_a += b1;
        j_a += b2;
      }
      else {
        i_a += b2;
        j_a += b1;
      }

      const CudaComplex<RealAlias<Scalar>> Ga =
          cond_conj(G_up[i_a + ldgu * j_a] + G_down[i_a + ldgd * j_a], conj_a);

      int w1_b(g4_helper.addWex(w2, w_ex));
      int w2_b(w2);
      int k1_b = g4_helper.addKex(k2, k_ex);
      int k2_b = k2;
      bool conj_b = false;
      if (g4_helper.get_bands() == 1)
        conj_b = g4_helper.extendGIndices(k1_b, k2_b, w1_b, w2_b);
      else
        conj_b = g4_helper.extendGIndicesMultiBand(k1_b, k2_b, w1_b, w2_b);

      int i_b = nb * k1_b + no * w1_b;
      int j_b = nb * k2_b + no * w2_b;
      if (conj_b) {
        i_b += b4;
        j_b += b3;
      }
      else {
        i_b += b3;
        j_b += b4;
      }

      const CudaComplex<RealAlias<Scalar>> Gb =
          cond_conj(G_up[i_b + ldgu * j_b] + G_down[i_b + ldgd * j_b], conj_b);

      contribution += sign_over_2 * (Ga * Gb);
    }
  }
  else if constexpr (type == FourPointType::PARTICLE_HOLE_LONGITUDINAL_UP_UP) {
    // The PARTICLE_HOLE_LONGITUDINAL_UP_UP contribution is computed in two parts:
    {
      // contribution <- \sum_s G(k1, k1+k_ex, s) * G(k2+k_ex, k2, s)
      int w1_a(w1);
      int w2_a(g4_helper.addWex(w1, w_ex));
      int k1_a = k1;
      int k2_a = g4_helper.addKex(k1, k_ex);
      bool conj_a = false;
      if (g4_helper.get_bands() == 1)
        conj_a = g4_helper.extendGIndices(k1_a, k2_a, w1_a, w2_a);
      else
        conj_a = g4_helper.extendGIndicesMultiBand(k1_a, k2_a, w1_a, w2_a);
      int i_a = nb * k1_a + no * w1_a;
      int j_a = nb * k2_a + no * w2_a;
      if (conj_a) {
        i_a += b4;
        j_a += b2;
      }
      else {
        i_a += b2;
        j_a += b4;
      }

      const CudaComplex<RealAlias<Scalar>> Ga_1 = cond_conj(G_up[i_a + ldgu * j_a], conj_a);
      const CudaComplex<RealAlias<Scalar>> Ga_2 = cond_conj(G_down[i_a + ldgd * j_a], conj_a);

      int w1_b(g4_helper.addWex(w2, w_ex));
      int w2_b(w2);
      int k1_b = g4_helper.addKex(k2, k_ex);
      int k2_b = k2;
      bool conj_b = false;
      if (g4_helper.get_bands() == 1)
        conj_b = g4_helper.extendGIndices(k1_b, k2_b, w1_b, w2_b);
      else
        conj_b = g4_helper.extendGIndicesMultiBand(k1_b, k2_b, w1_b, w2_b);

      int i_b = nb * k1_b + no * w1_b;
      int j_b = nb * k2_b + no * w2_b;
      if (conj_b) {
        i_b += b1;
        j_b += b3;
      }
      else {
        i_b += b3;
        j_b += b1;
      }

      const CudaComplex<RealAlias<Scalar>> Gb_1 = cond_conj(G_up[i_b + ldgd * j_b], conj_b);
      const CudaComplex<RealAlias<Scalar>> Gb_2 = cond_conj(G_down[i_b + ldgu * j_b], conj_b);

      contribution = sign_over_2 * (Ga_1 * Gb_1 + Ga_2 * Gb_2);
    }
    {
      // contribution <- -\sum_s G(k1, k2, s) * G(k2 + k_ex, k1 + k_ex, s)
      int w1_a(w1);
      int w2_a(w2);
      int k1_a(k1);
      int k2_a(k2);

      bool conj_a = false;
      if (g4_helper.get_bands() == 1)
        conj_a = g4_helper.extendGIndices(k1_a, k2_a, w1_a, w2_a);
      else
        conj_a = g4_helper.extendGIndicesMultiBand(k1_a, k2_a, w1_a, w2_a);
      int i_a = nb * k1_a + no * w1_a;
      int j_a = nb * k2_a + no * w2_a;
      if (conj_a) {
        i_a += b4;
        j_a += b1;
      }
      else {
        i_a += b1;
        j_a += b4;
      }
      const CudaComplex<RealAlias<Scalar>> Ga_1 = cond_conj(G_up[i_a + ldgu * j_a], conj_a);
      const CudaComplex<RealAlias<Scalar>> Ga_2 = cond_conj(G_down[i_a + ldgd * j_a], conj_a);

      int w1_b(g4_helper.addWex(w2, w_ex));
      int w2_b(g4_helper.addWex(w1, w_ex));
      int k1_b = g4_helper.addKex(k2, k_ex);
      int k2_b = g4_helper.addKex(k1, k_ex);

      bool conj_b = false;
      if (g4_helper.get_bands() == 1)
        conj_b = g4_helper.extendGIndices(k1_b, k2_b, w1_b, w2_b);
      else
        conj_b = g4_helper.extendGIndicesMultiBand(k1_b, k2_b, w1_b, w2_b);

      int i_b = nb * k1_b + no * w1_b;
      int j_b = nb * k2_b + no * w2_b;
      if (conj_b) {
        i_b += b3;
        j_b += b2;
      }
      else {
        i_b += b2;
        j_b += b3;
      }

      const CudaComplex<RealAlias<Scalar>> Gb_1 = cond_conj(G_up[i_b + ldgd * j_b], conj_b);
      const CudaComplex<RealAlias<Scalar>> Gb_2 = cond_conj(G_down[i_b + ldgu * j_b], conj_b);

      contribution += -sign_over_2 * (Ga_1 * Gb_1 + Ga_2 * Gb_2);
    }
  }
  else if constexpr (type == FourPointType::PARTICLE_HOLE_LONGITUDINAL_UP_DOWN) {
    // contribution <- \sum_s G(k1, k1+k_ex, s) * G(k2+k_ex, k2, -s)
    int w1_a(w1);
    int w2_a(g4_helper.addWex(w1, w_ex));
    int k1_a = k1;
    int k2_a = g4_helper.addKex(k1, k_ex);
    bool conj_a = false;
    if (g4_helper.get_bands() == 1)
      conj_a = g4_helper.extendGIndices(k1_a, k2_a, w1_a, w2_a);
    else
      conj_a = g4_helper.extendGIndicesMultiBand(k1_a, k2_a, w1_a, w2_a);
    int i_a = nb * k1_a + no * w1_a;
    int j_a = nb * k2_a + no * w2_a;
    if (conj_a) {
      i_a += b4;
      j_a += b2;
    }
    else {
      i_a += b2;
      j_a += b4;
    }

    const CudaComplex<RealAlias<Scalar>> Ga_1 = cond_conj(G_up[i_a + ldgu * j_a], conj_a);
    const CudaComplex<RealAlias<Scalar>> Ga_2 = cond_conj(G_down[i_a + ldgd * j_a], conj_a);

    int w1_b(g4_helper.addWex(w2, w_ex));
    int w2_b(w2);
    int k1_b = g4_helper.addKex(k2, k_ex);
    int k2_b = k2;
    bool conj_b = false;
    if (g4_helper.get_bands() == 1)
      conj_b = g4_helper.extendGIndices(k1_b, k2_b, w1_b, w2_b);
    else
      conj_b = g4_helper.extendGIndicesMultiBand(k1_b, k2_b, w1_b, w2_b);

    int i_b = nb * k1_b + no * w1_b;
    int j_b = nb * k2_b + no * w2_b;
    if (conj_b) {
      i_b += b1;
      j_b += b3;
    }
    else {
      i_b += b3;
      j_b += b1;
    }

    const CudaComplex<RealAlias<Scalar>> Gb_1 = cond_conj(G_down[i_b + ldgd * j_b], conj_b);
    const CudaComplex<RealAlias<Scalar>> Gb_2 = cond_conj(G_up[i_b + ldgu * j_b], conj_b);

    contribution = sign_over_2 * (Ga_1 * Gb_1 + Ga_2 * Gb_2);
  }
  else if constexpr (type == FourPointType::PARTICLE_PARTICLE_UP_DOWN) {
    // contribution <- -\sum_s G(k_ex - k2, k_ex - k1, s) * G(k2, k1, -s).
    int w1_a(w1);
    int w2_a(w2);
    int k1_a(k1);
    int k2_a(k2);
    bool conj_a = false;
    if (g4_helper.get_bands() == 1)
      conj_a = g4_helper.extendGIndices(k1_a, k2_a, w1_a, w2_a);
    else
      conj_a = g4_helper.extendGIndicesMultiBand(k1_a, k2_a, w1_a, w2_a);
    int i_a = nb * k1_a + no * w1_a;
    int j_a = nb * k2_a + no * w2_a;
    if (conj_a) {
      i_a += b4;
      j_a += b2;
    }
    else {
      i_a += b2;
      j_a += b4;
    }
    const CudaComplex<RealAlias<Scalar>> Ga_1 = cond_conj(G_up[i_a + ldgu * j_a], conj_a);
    const CudaComplex<RealAlias<Scalar>> Ga_2 = cond_conj(G_down[i_a + ldgd * j_a], conj_a);

    int w1_b(g4_helper.wexMinus(w1, w_ex));
    int w2_b(g4_helper.wexMinus(w2, w_ex));
    int k1_b = g4_helper.kexMinus(k1, k_ex);
    int k2_b = g4_helper.kexMinus(k2, k_ex);
    bool conj_b = false;
    if (g4_helper.get_bands() == 1)
      conj_b = g4_helper.extendGIndices(k1_b, k2_b, w1_b, w2_b);
    else
      conj_b = g4_helper.extendGIndicesMultiBand(k1_b, k2_b, w1_b, w2_b);

    int i_b = nb * k1_b + no * w1_b;
    int j_b = nb * k2_b + no * w2_b;
    if (conj_b) {
      i_b += b1;
      j_b += b3;
    }
    else {
      i_b += b3;
      j_b += b1;
    }

    const CudaComplex<RealAlias<Scalar>> Gb_1 = cond_conj(G_down[i_b + ldgd * j_b], conj_b);
    const CudaComplex<RealAlias<Scalar>> Gb_2 = cond_conj(G_up[i_b + ldgu * j_b], conj_b);

    contribution = sign_over_2 * (Ga_1 * Gb_1 + Ga_2 * Gb_2);
  }

  decltype(G4) const result_ptr = G4 + local_g4_index;
  if (atomic)
    dca::linalg::atomicAdd(result_ptr, contribution);
  else
    *result_ptr += contribution;
}

template <typename Scalar, FourPointType type, typename SignType>
double updateG4(Scalar* G4, const Scalar* G_up, const int ldgu, const Scalar* G_down,
                const int ldgd, const SignType factor, bool atomic, hipStream_t stream,
                std::size_t start, std::size_t end) {
  constexpr const std::size_t n_threads = 256;
  const unsigned n_blocks = dca::util::ceilDiv(end - start, n_threads);

  using dca::util::GPUTypeConversion;
  updateG4Kernel<dca::util::CUDATypeMap<Scalar>, type><<<n_blocks, n_threads, 0, stream>>>(
      castGPUType(G4), castGPUType(G_up), ldgu, castGPUType(G_down), ldgd,
      GPUTypeConversion(factor), atomic, start, end);

  // Check for errors.
  auto err = hipPeekAtLastError();
  if (err != hipSuccess) {
    linalg::util::printErrorMessage(err, __FUNCTION__, __FILE__, __LINE__);
    throw(std::runtime_error("CUDA failed to launch the G4 kernel."));
  }

  const std::size_t n_updates = end - start;
  switch (type) {
      // Note: sign flips  are ignored and a single complex * real multiplication is
      // present in all modes.
    case FourPointType::PARTICLE_HOLE_TRANSVERSE:
      // Each update of a G4 entry involves 2 complex additions and 2 complex multiplications.
      return 18. * n_updates;
    case FourPointType::PARTICLE_HOLE_MAGNETIC:
      // Each update of a G4 entry involves 3 complex additions and 3 complex multiplications.
      return 26. * n_updates;
    case FourPointType::PARTICLE_HOLE_CHARGE:
      // Each update of a G4 entry involves 3 complex additions and 3 complex multiplications.
      return 26. * n_updates;
    case FourPointType::PARTICLE_HOLE_LONGITUDINAL_UP_UP:
      // Each update of a G4 entry involves 3 complex additions and 4 complex multiplications.
      return 32 * n_updates;
    case FourPointType::PARTICLE_HOLE_LONGITUDINAL_UP_DOWN:
      // Each update of a G4 entry involves 2 complex additions and 2 complex multiplications.
      return 18. * n_updates;
    case FourPointType::PARTICLE_PARTICLE_UP_DOWN:
      // Each update of a G4 entry involves 2 complex additions and 2 complex multiplications.
      return 18. * n_updates;
    default:
      throw(std::logic_error("Invalid mode"));
  }
}

// Explicit instantiation.
template void computeGSingleband<float>(std::complex<float>* G, int ldg,
                                        const std::complex<float>* G0, int nk, int nw,
                                        const float beta, hipStream_t stream, int spin);
template void computeGMultiband<float>(std::complex<float>* G, int ldg,
                                       const std::complex<float>* G0, int ldg0, int nb, int nk,
                                       int nw, float beta, hipStream_t stream);

template void computeGSingleband<double>(std::complex<double>* G, int ldg,
                                         const std::complex<double>* G0, int nk, int nw_pos,
                                         const double beta, hipStream_t stream, int spin);
template void computeGMultiband<double>(std::complex<double>* G, int ldg,
                                        const std::complex<double>* G0, int ldg0, int nb, int nk,
                                        int nw_pos, double beta, hipStream_t stream);

template double updateG4<std::complex<float>, FourPointType::PARTICLE_HOLE_TRANSVERSE, std::int8_t>(
    std::complex<float>* G4, const std::complex<float>* G_up, const int ldgu,
    const std::complex<float>* G_down, const int ldgd, const std::int8_t factor, bool atomic,
    hipStream_t stream, std::size_t start, std::size_t end);

template double updateG4<std::complex<float>, FourPointType::PARTICLE_HOLE_MAGNETIC, std::int8_t>(
    std::complex<float>* G4, const std::complex<float>* G_up, const int ldgu,
    const std::complex<float>* G_down, const int ldgd, const std::int8_t factor, bool atomic,
    hipStream_t stream, std::size_t start, std::size_t end);

template double updateG4<std::complex<float>, FourPointType::PARTICLE_HOLE_CHARGE, std::int8_t>(
    std::complex<float>* G4, const std::complex<float>* G_up, const int ldgu,
    const std::complex<float>* G_down, const int ldgd, const std::int8_t factor, bool atomic,
    hipStream_t stream, std::size_t start, std::size_t end);

template double updateG4<std::complex<float>, FourPointType::PARTICLE_HOLE_LONGITUDINAL_UP_UP,
                         std::int8_t>(std::complex<float>* G4, const std::complex<float>* G_up,
                                      const int ldgu, const std::complex<float>* G_down,
                                      const int ldgd, const std::int8_t factor, bool atomic,
                                      hipStream_t stream, std::size_t start, std::size_t end);

template double updateG4<std::complex<float>, FourPointType::PARTICLE_HOLE_LONGITUDINAL_UP_DOWN,
                         std::int8_t>(std::complex<float>* G4, const std::complex<float>* G_up,
                                      const int ldgu, const std::complex<float>* G_down,
                                      const int ldgd, const std::int8_t factor, bool atomic,
                                      hipStream_t stream, std::size_t start, std::size_t end);

template double updateG4<std::complex<float>, FourPointType::PARTICLE_PARTICLE_UP_DOWN, std::int8_t>(
    std::complex<float>* G4, const std::complex<float>* G_up, const int ldgu,
    const std::complex<float>* G_down, const int ldgd, const std::int8_t factor, bool atomic,
    hipStream_t stream, std::size_t start, std::size_t end);

template double updateG4<std::complex<double>, FourPointType::PARTICLE_HOLE_TRANSVERSE, std::int8_t>(
    std::complex<double>* G4, const std::complex<double>* G_up, const int ldgu,
    const std::complex<double>* G_down, const int ldgd, const std::int8_t factor, bool atomic,
    hipStream_t stream, std::size_t start, std::size_t end);

template double updateG4<std::complex<double>, FourPointType::PARTICLE_HOLE_MAGNETIC, std::int8_t>(
    std::complex<double>* G4, const std::complex<double>* G_up, const int ldgu,
    const std::complex<double>* G_down, const int ldgd, const std::int8_t factor, bool atomic,
    hipStream_t stream, std::size_t start, std::size_t end);

template double updateG4<std::complex<double>, FourPointType::PARTICLE_HOLE_CHARGE, std::int8_t>(
    std::complex<double>* G4, const std::complex<double>* G_up, const int ldgu,
    const std::complex<double>* G_down, const int ldgd, const std::int8_t factor, bool atomic,
    hipStream_t stream, std::size_t start, std::size_t end);

template double updateG4<std::complex<double>, FourPointType::PARTICLE_HOLE_LONGITUDINAL_UP_UP,
                         std::int8_t>(std::complex<double>* G4, const std::complex<double>* G_up,
                                      const int ldgu, const std::complex<double>* G_down,
                                      const int ldgd, const std::int8_t factor, bool atomic,
                                      hipStream_t stream, std::size_t start, std::size_t end);

template double updateG4<std::complex<double>, FourPointType::PARTICLE_HOLE_LONGITUDINAL_UP_DOWN,
                         std::int8_t>(std::complex<double>* G4, const std::complex<double>* G_up,
                                      const int ldgu, const std::complex<double>* G_down,
                                      const int ldgd, const std::int8_t factor, bool atomic,
                                      hipStream_t stream, std::size_t start, std::size_t end);

template double updateG4<std::complex<double>, FourPointType::PARTICLE_PARTICLE_UP_DOWN, std::int8_t>(
    std::complex<double>* G4, const std::complex<double>* G_up, const int ldgu,
    const std::complex<double>* G_down, const int ldgd, const std::int8_t factor, bool atomic,
    hipStream_t stream, std::size_t start, std::size_t end);

// complex g0

template <>
double updateG4<std::complex<float>, FourPointType::PARTICLE_HOLE_TRANSVERSE, std::complex<float>>(
    std::complex<float>* G4, const std::complex<float>* G_up, const int ldgu,
    const std::complex<float>* G_down, const int ldgd, const std::complex<float> factor,
    bool atomic, hipStream_t stream, std::size_t start, std::size_t end);

template <>
double updateG4<std::complex<float>, FourPointType::PARTICLE_HOLE_MAGNETIC, std::complex<float>>(
    std::complex<float>* G4, const std::complex<float>* G_up, const int ldgu,
    const std::complex<float>* G_down, const int ldgd, const std::complex<float> factor,
    bool atomic, hipStream_t stream, std::size_t start, std::size_t end);

template <>
double updateG4<std::complex<float>, FourPointType::PARTICLE_HOLE_CHARGE, std::complex<float>>(
    std::complex<float>* G4, const std::complex<float>* G_up, const int ldgu,
    const std::complex<float>* G_down, const int ldgd, const std::complex<float> factor,
    bool atomic, hipStream_t stream, std::size_t start, std::size_t end);

template <>
double updateG4<std::complex<float>, FourPointType::PARTICLE_HOLE_LONGITUDINAL_UP_UP,
                std::complex<float>>(std::complex<float>* G4, const std::complex<float>* G_up,
                                     const int ldgu, const std::complex<float>* G_down,
                                     const int ldgd, const std::complex<float> factor, bool atomic,
                                     hipStream_t stream, std::size_t start, std::size_t end);

template <>
double updateG4<std::complex<float>, FourPointType::PARTICLE_HOLE_LONGITUDINAL_UP_DOWN,
                std::complex<float>>(std::complex<float>* G4, const std::complex<float>* G_up,
                                     const int ldgu, const std::complex<float>* G_down,
                                     const int ldgd, const std::complex<float> factor, bool atomic,
                                     hipStream_t stream, std::size_t start, std::size_t end);

template <>
double updateG4<std::complex<float>, FourPointType::PARTICLE_PARTICLE_UP_DOWN, std::complex<float>>(
    std::complex<float>* G4, const std::complex<float>* G_up, const int ldgu,
    const std::complex<float>* G_down, const int ldgd, const std::complex<float> factor,
    bool atomic, hipStream_t stream, std::size_t start, std::size_t end);

template <>
double updateG4<std::complex<double>, FourPointType::PARTICLE_HOLE_TRANSVERSE, std::complex<double>>(
    std::complex<double>* G4, const std::complex<double>* G_up, const int ldgu,
    const std::complex<double>* G_down, const int ldgd, const std::complex<double> factor,
    bool atomic, hipStream_t stream, std::size_t start, std::size_t end);

template <>
double updateG4<std::complex<double>, FourPointType::PARTICLE_HOLE_MAGNETIC, std::complex<double>>(
    std::complex<double>* G4, const std::complex<double>* G_up, const int ldgu,
    const std::complex<double>* G_down, const int ldgd, const std::complex<double> factor,
    bool atomic, hipStream_t stream, std::size_t start, std::size_t end);

template <>
double updateG4<std::complex<double>, FourPointType::PARTICLE_HOLE_CHARGE, std::complex<double>>(
    std::complex<double>* G4, const std::complex<double>* G_up, const int ldgu,
    const std::complex<double>* G_down, const int ldgd, const std::complex<double> factor,
    bool atomic, hipStream_t stream, std::size_t start, std::size_t end);

template <>
double updateG4<std::complex<double>, FourPointType::PARTICLE_HOLE_LONGITUDINAL_UP_UP,
                std::complex<double>>(std::complex<double>* G4, const std::complex<double>* G_up,
                                      const int ldgu, const std::complex<double>* G_down,
                                      const int ldgd, const std::complex<double> factor, bool atomic,
                                      hipStream_t stream, std::size_t start, std::size_t end);

template <>
double updateG4<std::complex<double>, FourPointType::PARTICLE_HOLE_LONGITUDINAL_UP_DOWN,
                std::complex<double>>(std::complex<double>* G4, const std::complex<double>* G_up,
                                      const int ldgu, const std::complex<double>* G_down,
                                      const int ldgd, const std::complex<double> factor, bool atomic,
                                      hipStream_t stream, std::size_t start, std::size_t end);

template <>
double updateG4<std::complex<double>, FourPointType::PARTICLE_PARTICLE_UP_DOWN, std::complex<double>>(
    std::complex<double>* G4, const std::complex<double>* G_up, const int ldgu,
    const std::complex<double>* G_down, const int ldgd, const std::complex<double> factor,
    bool atomic, hipStream_t stream, std::size_t start, std::size_t end);

// template<> double updateG4< FourPointType::PARTICLE_HOLE_TRANSVERSE>(
//   std::complex<float>* G4, const std::complex<float>* G_up, const int ldgu,
//   const std::complex<float>* G_down, const int ldgd, const std::int8_t factor, bool atomic,
//   hipStream_t stream, std::size_t start, std::size_t end);

// template<> double updateG4< FourPointType::PARTICLE_HOLE_MAGNETIC>(
//   std::complex<float>* G4, const std::complex<float>* G_up, const int ldgu,
//   const std::complex<float>* G_down, const int ldgd, const std::int8_t factor, bool atomic,
//   hipStream_t stream, std::size_t start, std::size_t end);

// template<> double updateG4< FourPointType::PARTICLE_HOLE_CHARGE>(
//   std::complex<float>* G4, const std::complex<float>* G_up, const int ldgu,
//   const std::complex<float>* G_down, const int ldgd, const std::int8_t factor, bool atomic,
//   hipStream_t stream, std::size_t start, std::size_t end);

// template<> double updateG4< FourPointType::PARTICLE_HOLE_LONGITUDINAL_UP_UP>(
//   std::complex<float>* G4, const std::complex<float>* G_up, const int ldgu,
//   const std::complex<float>* G_down, const int ldgd, const std::int8_t factor, bool atomic,
//   hipStream_t stream, std::size_t start, std::size_t end);

// template<> double updateG4< FourPointType::PARTICLE_HOLE_LONGITUDINAL_UP_DOWN>(
//   std::complex<float>* G4, const std::complex<float>* G_up, const int ldgu,
//   const std::complex<float>* G_down, const int ldgd, const std::int8_t factor, bool atomic,
//   hipStream_t stream, std::size_t start, std::size_t end);

// template<> double updateG4< FourPointType::PARTICLE_PARTICLE_UP_DOWN>(
//   std::complex<float>* G4, const std::complex<float>* G_up, const int ldgu,
//   const std::complex<float>* G_down, const int ldgd, const std::int8_t factor, bool atomic,
//   hipStream_t stream, std::size_t start, std::size_t end);

// template<> double updateG4< FourPointType::PARTICLE_HOLE_TRANSVERSE>(
//   std::complex<double>* G4, const std::complex<double>* G_up, const int ldgu,
//   const std::complex<double>* G_down, const int ldgd, const std::int8_t factor, bool atomic,
//   hipStream_t stream, std::size_t start, std::size_t end);

// template<> double updateG4< FourPointType::PARTICLE_HOLE_MAGNETIC>(
//   std::complex<double>* G4, const std::complex<double>* G_up, const int ldgu,
//   const std::complex<double>* G_down, const int ldgd, const std::int8_t factor, bool atomic,
//   hipStream_t stream, std::size_t start, std::size_t end);

// template<> double updateG4< FourPointType::PARTICLE_HOLE_CHARGE>(
//   std::complex<double>* G4, const std::complex<double>* G_up, const int ldgu,
//   const std::complex<double>* G_down, const int ldgd, const std::int8_t factor, bool atomic,
//   hipStream_t stream, std::size_t start, std::size_t end);

// template<> double updateG4< FourPointType::PARTICLE_HOLE_LONGITUDINAL_UP_UP>(
//   std::complex<double>* G4, const std::complex<double>* G_up, const int ldgu,
//   const std::complex<double>* G_down, const int ldgd, const std::int8_t factor, bool atomic,
//   hipStream_t stream, std::size_t start, std::size_t end);

// template<> double updateG4< FourPointType::PARTICLE_HOLE_LONGITUDINAL_UP_DOWN>(
//   std::complex<double>* G4, const std::complex<double>* G_up, const int ldgu,
//   const std::complex<double>* G_down, const int ldgd, const std::int8_t factor, bool atomic,
//   hipStream_t stream, std::size_t start, std::size_t end);

// template<> double updateG4< FourPointType::PARTICLE_PARTICLE_UP_DOWN>(
//   std::complex<double>* G4, const std::complex<double>* G_up, const int ldgu,
//   const std::complex<double>* G_down, const int ldgd, const std::int8_t factor, bool atomic,
//   hipStream_t stream, std::size_t start, std::size_t end);

}  // namespace details
}  // namespace accumulator
}  // namespace solver
}  // namespace phys
}  // namespace dca
