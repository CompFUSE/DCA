#include "hip/hip_runtime.h"
// Copyright (C) 2019 ETH Zurich
// Copyright (C) 2019 UT-Battelle, LLC
// All rights reserved.
//
// See LICENSE for terms of usage.
// See CITATION.md for citation guidelines, if DCA++ is used for scientific publications.
//
// Author: Giovanni Balduzzi (gbalduzz@itp.phys.ethz.ch)
//
// Implementation of the G0 computation for time measurements.

#include "dca/phys/dca_step/cluster_solver/shared_tools/accumulation/kernels_interface.hpp"

#include "dca/linalg/util/cast_cuda.hpp"
#include "dca/util/cuda_blocks.hpp"
#include "dca/phys/dca_step/cluster_solver/shared_tools/solver_helper.cuh"

namespace dca {
namespace phys {
namespace solver {
namespace details {
// dca::phys::solver::details::

using namespace dca::linalg;

template <typename Scalar, typename Real>
__global__ void computeG0Kernel(linalg::MatrixView<Scalar, linalg::GPU> mat,
                                const DeviceInterpolationData<Scalar> g0, const Real* t_l,
                                const int* b_l, const int* r_l, const Real* t_r, const int* b_r,
                                const int* r_r) {
  const unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned j = blockIdx.y * blockDim.y + threadIdx.y;
  if (i >= mat.nrRows() || j >= mat.nrCols())
    return;

  const auto index = solver_helper.index(b_l[i], b_r[j], r_l[i], r_r[j]);
  const Real tau = t_l[i] - t_r[j];

  castCuda(mat(i, j)) = castCuda(g0(tau, index));
}

template <class T>
struct CudaScalarImpl {
  using type = T;
};
template <>
struct CudaScalarImpl<std::complex<double>> {
  using type = hipDoubleComplex;
};
template <>
struct CudaScalarImpl<std::complex<float>> {
  using type = hipComplex;
};

template <typename Scalar, typename Real>
void computeG0(linalg::MatrixView<Scalar, linalg::GPU>& g0_mat,
               const DeviceInterpolationData<Scalar> g0, const Real* t_l, const int* b_l,
               const int* r_l, const Real* t_r, const int* b_r, const int* r_r, hipStream_t stream) {
  assert(SolverHelper::initialized());
  auto blocks = dca::util::get2DBlockSize(g0_mat.nrRows(), g0_mat.nrCols(), 32);
  using dca::linalg::castCuda;

  computeG0Kernel<<<blocks[0], blocks[1], 0, stream>>>(g0_mat, g0, t_l, b_l, r_l, t_r, b_r, r_r);
}

// Instantation.
template void computeG0(linalg::MatrixView<double, linalg::GPU>&,
                        const DeviceInterpolationData<double>, const double*, const int*,
                        const int*, const double*, const int*, const int*, hipStream_t);
template void computeG0(linalg::MatrixView<float, linalg::GPU>&,
                        const DeviceInterpolationData<float>, const float*, const int*, const int*,
                        const float*, const int*, const int*, hipStream_t);
template void computeG0(linalg::MatrixView<std::complex<double>, linalg::GPU>&,
                        const DeviceInterpolationData<std::complex<double>>, const double*,
                        const int*, const int*, const double*, const int*, const int*, hipStream_t);
template void computeG0(linalg::MatrixView<std::complex<float>, linalg::GPU>&,
                        const DeviceInterpolationData<std::complex<float>>, const float*,
                        const int*, const int*, const float*, const int*, const int*, hipStream_t);

}  // namespace details
}  // namespace solver
}  // namespace phys
}  // namespace dca
