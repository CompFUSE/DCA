#include "hip/hip_runtime.h"
// Copyright (C) 2018 ETH Zurich
// Copyright (C) 2018 UT-Battelle, LLC
// All rights reserved.
//
// See LICENSE for terms of usage.
// See CITATION.md for citation guidelines, if DCA++ is used for scientific publications.
//
// Author: Peter Staar (taa@zurich.ibm.com)
//
// This file implements ct_aux_walker_tools_kernels.hpp.

#include "dca/phys/dca_step/cluster_solver/ctaux/walker/ct_aux_walker_tools_kernels.hpp"

#include "hip/hip_runtime.h"

#include "dca/linalg/util/error_cuda.hpp"
#include "dca/linalg/util/stream_functions.hpp"
#include "dca/util/integer_division.hpp"

namespace dca {
namespace phys {
namespace solver {
namespace ctaux {
namespace walkerkernels {
// dca::phys::solver::ctaux::walkerkernels::

template <typename Real>
__global__ void compute_Gamma_kernel(Real* Gamma, int Gamma_n, int Gamma_ld, Real* N, int N_r,
                                     int N_c, int N_ld, Real* G, int G_r, int G_c, int G_ld,
                                     int* random_vertex_vector, Real* exp_V, Real* exp_delta_V) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  int vertex_index = N_c - G_c;

  if (i < Gamma_n and j < Gamma_n) {
    int configuration_e_spin_index_i = random_vertex_vector[i];
    int configuration_e_spin_index_j = random_vertex_vector[j];

    if (configuration_e_spin_index_j < vertex_index) {
      Real delta = 0;

      if (configuration_e_spin_index_i == configuration_e_spin_index_j)
        delta = 1.;

      Real N_ij = N[configuration_e_spin_index_i + configuration_e_spin_index_j * N_ld];

      Gamma[i + j * Gamma_ld] = (N_ij * exp_V[j] - delta) / (exp_V[j] - 1.);
    }
    else
      Gamma[i + j * Gamma_ld] =
          G[configuration_e_spin_index_i + (configuration_e_spin_index_j - vertex_index) * G_ld];
  }

  if (i < Gamma_n and j < Gamma_n and i == j) {
    Real gamma_k = exp_delta_V[j];
    Gamma[i + j * Gamma_ld] -= (gamma_k) / (gamma_k - 1.);
  }
}

template <typename Real>
void compute_Gamma(Real* Gamma, int Gamma_n, int Gamma_ld, Real* N, int N_r, int N_c, int N_ld,
                   Real* G, int G_r, int G_c, int G_ld, int* random_vertex_vector, Real* exp_V,
                   Real* exp_delta_V, int thread_id, int stream_id) {
  const int number_of_threads = 16;

  if (Gamma_n > 0) {
    checkErrorsCudaDebug();

    dim3 threads(number_of_threads, number_of_threads);

    dim3 blocks(dca::util::ceilDiv(Gamma_n, number_of_threads),
                dca::util::ceilDiv(Gamma_n, number_of_threads));

    hipStream_t stream_handle = dca::linalg::util::getStream(thread_id, stream_id);

    compute_Gamma_kernel<<<blocks, threads, 0, stream_handle>>>(
        Gamma, Gamma_n, Gamma_ld, N, N_r, N_c, N_ld, G, G_r, G_c, G_ld, random_vertex_vector, exp_V,
        exp_delta_V);

    checkErrorsCudaDebug();
  }
}
template void compute_Gamma(float* Gamma, int Gamma_n, int Gamma_ld, float* N, int N_r, int N_c,
                            int N_ld, float* G, int G_r, int G_c, int G_ld, int* random_vertex_vector,
                            float* exp_V, float* exp_delta_V, int thread_id, int stream_id);
template void compute_Gamma(double* Gamma, int Gamma_n, int Gamma_ld, double* N, int N_r, int N_c,
                            int N_ld, double* G, int G_r, int G_c, int G_ld,
                            int* random_vertex_vector, double* exp_V, double* exp_delta_V,
                            int thread_id, int stream_id);

}  // namespace walkerkernels
}  // namespace ctaux
}  // namespace solver
}  // namespace phys
}  // namespace dca
