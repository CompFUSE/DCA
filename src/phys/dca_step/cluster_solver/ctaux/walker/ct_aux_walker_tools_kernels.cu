#include "hip/hip_runtime.h"
// Copyright (C) 2018 ETH Zurich
// Copyright (C) 2018 UT-Battelle, LLC
// All rights reserved.
//
// See LICENSE for terms of usage.
// See CITATION.md for citation guidelines, if DCA++ is used for scientific publications.
//
// Author: Peter Staar (taa@zurich.ibm.com)
//
// This file implements ct_aux_walker_tools_kernels.hpp.

#include "dca/phys/dca_step/cluster_solver/ctaux/walker/ct_aux_walker_tools_kernels.hpp"

#include <type_traits>

#include "dca/platform/dca_gpu.h"
#include "dca/util/type_help.hpp"
#include "dca/linalg/util/complex_operators_cuda.cu.hpp"

#include "dca/linalg/util/stream_functions.hpp"
#include "dca/util/integer_division.hpp"
#include "dca/util/type_help.hpp"

namespace dca {
namespace phys {
namespace solver {
namespace ctaux {
namespace walkerkernels {
// dca::phys::solver::ctaux::walkerkernels::

  template<typename T>
  using IsCudaComplex_t = dca::util::IsCudaComplex_t<T>;
  
template <class T>
__global__ void compute_Gamma_kernel(T* Gamma, int Gamma_n, int Gamma_ld, const T* N, int N_r,
                                     int N_c, int N_ld, const T* G, int G_r, int G_c, int G_ld,
                                     const int* random_vertex_vector, const T* exp_V,
                                     const T* exp_delta_V) {
  using namespace dca::linalg;

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  const int vertex_index = N_c - G_c;

    T the_one;
    T the_zero{};
    if constexpr (IsCudaComplex_t<T>::value)
      the_one = T{1.0, 0.0};
    else
      the_one = 1.0;
      
  if (i < Gamma_n and j < Gamma_n) {
    const int configuration_e_spin_index_i = random_vertex_vector[i];
    const int configuration_e_spin_index_j = random_vertex_vector[j];

    if (configuration_e_spin_index_j < vertex_index) {
      T delta;
      if (configuration_e_spin_index_i == configuration_e_spin_index_j)
        if constexpr (dca::util::IsCudaComplex_t<T>::value)
          delta = {1., 0};
        else
          delta = 1;

      const auto N_ij = N[configuration_e_spin_index_i + configuration_e_spin_index_j * N_ld];

      
      
      Gamma[i + j * Gamma_ld] = (N_ij * exp_V[j] - delta) / (exp_V[j] - the_one);
    }
    else
      Gamma[i + j * Gamma_ld] =
          G[configuration_e_spin_index_i + (configuration_e_spin_index_j - vertex_index) * G_ld];
  }

  if (i < Gamma_n and j < Gamma_n and i == j) {
    const auto gamma_k = exp_delta_V[j];
    Gamma[i + j * Gamma_ld] -= (gamma_k) / (gamma_k - the_one);
  }
}
    
template <class T>
void compute_Gamma(T* Gamma, int Gamma_n, int Gamma_ld, const T* N, int N_r, int N_c, int N_ld,
                   const T* G, int G_r, int G_c, int G_ld, const int* random_vertex_vector,
                   const T* exp_V, const T* exp_delta_V, int thread_id, int stream_id) {
  const int number_of_threads = 16;

  if (Gamma_n > 0) {
    checkErrorsCudaDebug();

    dim3 threads(number_of_threads, number_of_threads);

    dim3 blocks(dca::util::ceilDiv(Gamma_n, number_of_threads),
                dca::util::ceilDiv(Gamma_n, number_of_threads));

    hipStream_t stream_handle = dca::linalg::util::getStream(thread_id, stream_id);

    using dca::util::castGPUType;
    compute_Gamma_kernel<<<blocks, threads, 0, stream_handle>>>(
        castGPUType(Gamma), Gamma_n, Gamma_ld, castGPUType(N), N_r, N_c, N_ld, castGPUType(G), G_r,
        G_c, G_ld, random_vertex_vector, castGPUType(exp_V), castGPUType(exp_delta_V));

    checkErrorsCudaDebug();
  }
}

template void compute_Gamma(float*, int, int, const float*, int, int, int, const float*, int, int,
                            int, const int*, const float*, const float*, int, int);
template void compute_Gamma(double*, int, int, const double*, int, int, int, const double*, int,
                            int, int, const int*, const double*, const double*, int, int);
template void compute_Gamma(std::complex<float>*, int, int, const std::complex<float>*, int, int,
                            int, const std::complex<float>*, int, int, int, const int*,
                            const std::complex<float>*, const std::complex<float>*, int, int);
template void compute_Gamma(std::complex<double>*, int, int, const std::complex<double>*, int, int,
                            int, const std::complex<double>*, int, int, int, const int*,
                            const std::complex<double>*, const std::complex<double>*, int, int);

}  // namespace walkerkernels
}  // namespace ctaux
}  // namespace solver
}  // namespace phys
}  // namespace dca
