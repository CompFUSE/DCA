#include "hip/hip_runtime.h"
// Copyright (C) 2018 ETH Zurich
// Copyright (C) 2018 UT-Battelle, LLC
// All rights reserved.
//
// See LICENSE.txt for terms of usage.
//  See CITATION.md for citation guidelines, if DCA++ is used for scientific publications.
//
// Author: Giovanni Balduzzi (gbalduzz@itp.phys.ethz.ch)
//
// Implementation of the device methods used by the CT-INT submatrix walker.

#include "dca/phys/dca_step/cluster_solver/ctint/walker/kernels_interface.hpp"

#include <array>
#include <cassert>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "dca/util/cuda_blocks.hpp"

namespace dca {
namespace phys {
namespace solver {
namespace ctint {
namespace details {

template <typename Real>
__global__ void setRightSectorToIdKernel(Real* m, const int ldm, const int n0, const int n_max) {
  const int i = threadIdx.x + blockDim.x * blockIdx.x;
  const int j = threadIdx.y + blockDim.y * blockIdx.y + n0;

  if (i >= n_max || j >= n_max)
    return;

  m[i + ldm * j] = (i == j) ? 1. : 0.;
}

template <typename Real>
void setRightSectorToId(Real* m, const int ldm, const int n0, const int n_max, hipStream_t stream) {
  auto blocks = dca::util::getBlockSize(n_max, n_max - n0);

  setRightSectorToIdKernel<<<blocks[0], blocks[1], 0, stream>>>(m, ldm, n0, n_max);
}

template void setRightSectorToId(float*, const int, const int, const int, hipStream_t);
template void setRightSectorToId(double*, const int, const int, const int, hipStream_t);

template <typename Real>
__global__ void computeGLeftKernel(MatrixView<Real> G, const MatrixView<Real> M,
                                   const Real* __restrict__ f, int n_init) {
  const int i_t = threadIdx.x + blockDim.x * blockIdx.x;
  const int stride = blockDim.x * gridDim.x;
  const int j = threadIdx.y + blockDim.y * blockIdx.y;
  if (j >= n_init)
    return;

  const Real factor = 1. / (f[j] - 1);
  const Real fj = f[j];

  for (int i = i_t; i < G.nrRows(); i += stride)
    G(i, j) = (M(i, j) * fj - Real(i == j)) * factor;
}

template <typename Real>
void computeGLeft(MatrixView<Real>& G, const MatrixView<Real>& M, const Real* f, int n_init,
                  hipStream_t stream) {
  if (n_init == 0)
    return;
  const int n = G.nrRows();

  constexpr int thread_j = 4;
  constexpr int thread_i = 64;
  dim3 threads(thread_i, thread_j);
  dim3 blocks(std::max(n / (10 * thread_i), 1), util::ceilDiv(n_init, thread_j));

  computeGLeftKernel<<<blocks, threads, 0, stream>>>(G, M, f, n_init);
}

template void computeGLeft(MatrixView<float>&, const MatrixView<float>&, const float*, int,
                           hipStream_t);
template void computeGLeft(MatrixView<double>&, const MatrixView<double>&, const double*, int,
                           hipStream_t);

template <typename Real>
__global__ void multiplyByFColFactorKernel(MatrixView<Real> M, const Real* f_vals) {
  const int i = threadIdx.x + blockDim.x * blockIdx.x;
  const int j = threadIdx.y + blockDim.y * blockIdx.y;
  if (i >= M.nrRows() || j >= M.nrCols())
    return;

  const Real factor = f_vals[j] - 1.;
  M(i, j) *= factor;
}

template <typename Real>
void multiplyByFColFactor(MatrixView<Real>& M, const Real* f_vals, hipStream_t stream) {
  if (M.nrCols() == 0 || M.nrRows() == 0)
    return;
  const auto blocks = dca::util::getBlockSize(M.nrRows(), M.nrCols());

  multiplyByFColFactorKernel<<<blocks[0], blocks[1], 0, stream>>>(M, f_vals);
}

template void multiplyByFColFactor(MatrixView<float>&, const float*, hipStream_t);
template void multiplyByFColFactor(MatrixView<double>&, const double*, hipStream_t);

template <typename Real>
__global__ void multiplyByInverseFFactorKernel(const MatrixView<Real> m_in, MatrixView<Real> m_out,
                                               const Real* f_vals) {
  const int i = threadIdx.x + blockDim.x * blockIdx.x;
  const int j = threadIdx.y + blockDim.y * blockIdx.y;
  if (i >= m_in.nrRows() || j >= m_in.nrCols())
    return;

  const Real factor = -(f_vals[i] - 1.);
  m_out(i, j) = factor * m_in(i, j);
}

template <typename Real>
void multiplyByInverseFFactor(const MatrixView<Real>& m_in, MatrixView<Real>& m_out,
                              const Real* f_vals, hipStream_t stream) {
  assert(m_in.nrRows() == m_out.nrRows() && m_in.nrCols() == m_out.nrCols());
  if (m_in.nrCols() == 0 || m_in.nrRows() == 0)
    return;
  const auto blocks = dca::util::getBlockSize(m_in.nrRows(), m_out.nrCols());

  multiplyByInverseFFactorKernel<<<blocks[0], blocks[1], 0, stream>>>(m_in, m_out, f_vals);
}

template void multiplyByInverseFFactor(const MatrixView<float>&, MatrixView<float>&, const float*,
                                       hipStream_t);
template void multiplyByInverseFFactor(const MatrixView<double>&, MatrixView<double>&,
                                       const double*, hipStream_t);

template <typename Real>
__global__ void divideByGammaFactorKernel(MatrixView<Real> m,
                                          const std::pair<int, Real>* gamma_indices,
                                          const int n_indices) {
  // TODO: loop over a number of j indices.
  const int i = threadIdx.x + blockDim.x * blockIdx.x;
  const int j = threadIdx.y + blockDim.y * blockIdx.y;
  if (i >= n_indices || j >= m.nrCols())
    return;

  const int p = gamma_indices[i].first;
  assert(p < m.nrRows());

  m(p, j) /= 1. + gamma_indices[i].second;
}

template <typename Real>
void divideByGammaFactor(MatrixView<Real> m, const std::pair<int, Real>* gamma_indices,
                         const int n_indices, hipStream_t stream) {
  const auto blocks = dca::util::getBlockSize(n_indices, m.nrCols());

  divideByGammaFactorKernel<<<blocks[0], blocks[1], 0, stream>>>(m, gamma_indices, n_indices);
}

template void divideByGammaFactor(MatrixView<float>, const std::pair<int, float>*, const int,
                                  hipStream_t);
template void divideByGammaFactor(MatrixView<double>, const std::pair<int, double>*, const int,
                                  hipStream_t);

}  // namespace details
}  // namespace ctint
}  // namespace solver
}  // namespace phys
}  // namespace dca
