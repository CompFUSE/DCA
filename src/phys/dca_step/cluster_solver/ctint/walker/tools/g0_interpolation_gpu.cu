#include "hip/hip_runtime.h"
// Copyright (C) 2018 ETH Zurich
// Copyright (C) 2018 UT-Battelle, LLC
// All rights reserved.
//
// See LICENSE.txt for terms of usage.
// See CITATION.md for citation guidelines, if DCA++ is used for scientific publications.
//
// Authors: Giovanni Balduzzi (gbalduzz@itp.phys.ethz.ch)
//
// This file implements the device methods of G0Interpolation<GPU>.

#include "dca/phys/dca_step/cluster_solver/ctint/walker/tools/g0_interpolation_gpu.hpp"

#include <hip/hip_runtime.h>

#include "dca/linalg/util/error_cuda.hpp"
#include "dca/util/cuda_blocks.hpp"

namespace dca {
namespace phys {
namespace solver {
namespace ctint {
// dca::phys::solver::ctint::

template <typename Real>
__device__ Real DeviceInterpolationData<Real>::operator()(Real tau, int lindex) const {
  assert(tau >= -beta_ && tau <= beta_);

  if (tau == 0)  // returns G0(tau = 0+)
    return g0_minus_[lindex];

  short int factor = 1;
  if (tau < 0) {
    tau += beta_;
    factor = -1;
  }

  // Scale tau in [0, n_time_slices). Assume even spacing in time.
  const Real scaled_tau = tau * n_div_beta_;
  const int tau_index(scaled_tau);
  const Real delta_tau = scaled_tau - tau_index;

  // Get the pointer to the first akima coeff.
  const Real* coeff_ptr = &values_[tau_index * coeff_size_ + lindex * stride_];

  // Return akima interpolation.
  return factor *
         (coeff_ptr[0] +
          delta_tau * (coeff_ptr[1] + delta_tau * (coeff_ptr[2] + delta_tau * coeff_ptr[3])));
}

template <typename Real>
__global__ void g0InterpolationTestKernel(Real tau, const int lindex,
                                          DeviceInterpolationData<Real> g0, Real* result) {
  *result = g0(tau, lindex);
}

template <typename Real>
Real G0Interpolation<linalg::GPU, Real>::operator()(Real tau, int lindex) const {
  Real* d_result;
  Real result;
  hipMalloc((void**)&d_result, sizeof(Real));

  g0InterpolationTestKernel<<<1, 1>>>(tau, lindex, *this, d_result);

  assert(hipSuccess == hipPeekAtLastError());
  hipMemcpy(&result, d_result, sizeof(Real), hipMemcpyDeviceToHost);
  hipFree(d_result);
  return result;
}

}  // namespace ctint
}  // namespace solver
}  // namespace phys
}  // namespace dca
